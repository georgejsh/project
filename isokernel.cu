#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include ""
#include <thrust/scan.h>
const int BASE1 = 10000 + 7;
const int BASE2 = 100000 + 3;
const int MOD1 = 1000000 + 3;
const int MOD2 = 1000000 + 37;
__global__ void findhash(int *d_qvert,int *d_qverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_qtree,int *d_hash1,int *d_hash2)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	int l=d_qvert[ver+1];
	int hash1=1,hash2=1;
	int flag=0;
	for(i=d_qvert[ver];i<l;i++)
	{
		int m=d_qelist[i];
		bool treeedge=d_qtree[i];
		if(treeedge){
			int tt=d_qvid[m];
			if(tt==0)
				return;
			flag=1;
			hash1=(hash1*1L*BASE1)*tt % MOD1;
			hash2=(hash2*1L*BASE2)*tt % MOD2;
		}
	}
	if(flag==0)
		return;
	if(flag==1){
		*d_over=false;
		d_hash1[hash1]=1;
		d_hash2[hash2]=1;
	}
	
}
__global__ void setdeg1(int *d_qvert,int *d_qverc,int *d_qvid,bool *d_qtree)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	
	int l=d_qvert[ver+1];
	bool treeedge;
	
	for(i=d_qvert[ver];i<l;i++)
	{
		treeedge=d_qtree[i];
		
		if(treeedge)
			return;
	}
//printf("%d %d\n",ver,i);
	d_qvid[ver]=1;	
}
/*__global__ void alignhash(bool *d_hash1,bool *d_hash2)
{
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=1000038)
		return ;
	if(d_hash1[ver] || d_hash2[ver]){
		d_hash1=true;
	}
}*/
__global__ void puttoid(int *d_qvert,int *d_qverc,int *d_qvid,int *d_qelist,bool *d_qtree,int *d_loc,int * d_qidtov)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	int l=d_qvert[ver+1];
	int hash1=1,hash2=1;
	int flag=0;
	for(i=d_qvert[ver];i<l;i++)
	{
		int m=d_qelist[i];
		bool treeedge=d_qtree[i];
		if(treeedge){
			int tt=d_qvid[m];
			if(tt==0)
				return;
			flag=1;
			hash1=(hash1*1L*BASE1)*tt % MOD1;
			hash2=(hash2*1L*BASE2)*tt % MOD2;
		}
	}
	//printf("%d %d %d \n",ver,flag,d_loc[hash1]);
	if(flag==0)
		return;
	int id=d_loc[hash1];
	d_qvid[ver]=id;
	d_qidtov[id]=ver;
}
__device__ bool chechall(int ver,bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,int ** d_dcvslist){
	//int ql=qfrom-qto;
	int ql=qto-qfrom;
	int j,k,l;
	//d_dcvslist[2][ql]=true;
	if(i==ql){
		
		k=d_qelist[i+qfrom-1];
		k=d_qvid[k];
		if(k>=d_qvid[ver])
			return true;
		for(j=dfrom;j<dto;j++){
			l=d_delist[j];
			if(check[j])
				continue;
			if(!d_dcvslist[k][l])
				continue;
			return true;
		}
	}
	else{
		int res=false;
		k=d_qelist[i+qfrom-1];
		k=d_qvid[k];
		if(k>=d_qvid[ver])
			return chechall(ver,check,i+1,dfrom,dto,d_delist,d_qelist,d_qvid,qfrom,qto,d_dcvslist);
		
		for(j=dfrom;j<dto;j++){
			l=d_delist[j];
			if(check[j])
				continue;
			if(!d_dcvslist[k][l])
				continue;
			check[j]=true;
			res|=chechall(ver,check,i+1,dfrom,dto,d_delist,d_qelist,d_qvid,qfrom,qto,d_dcvslist);
			if(res==true)
				return true;
			check[j]=false;
		}
	}
	return false;
}
__global__ void findcvs(bool *temp,int ver,int *d_dvert,int *d_dverc,int *d_delist,int *d_qvert,int *d_qelist,int *d_qvid,int ** d_dcvslist )
{
	//int i;
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	int ql=d_qvert[ver+1]-d_qvert[ver];
	int dl=d_dvert[dver+1]-d_dvert[dver];
	if(ql>dl)
		return;
	//bool *checked=(bool*)malloc(sizeof(bool)*d_dverc[0]);
	//bool *checked=new bool[d_dverc[0]];
	//memset(checked,false,sizeof(bool)*d_dverc[0]);
	//chechall(bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,bool ** d_dcvslist)
	if(chechall(ver,temp,1,d_dvert[dver],d_dvert[dver+1],d_delist,d_qelist,d_qvid,d_qvert[ver],d_qvert[ver+1],d_dcvslist))
		d_dcvslist[d_qvid[ver]][dver]=true;
	//free(checked);
}
__global__ void puttolist(int *d_dverc,int *d_loc,int * d_dcvslist )
{
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	if(d_loc[dver]!=d_loc[dver+1])
		d_dcvslist[d_loc[dver]]=dver;
}
__global__ void checkperm(int *found,int * qdmap,int * d_qverc,int * d_qelist,int * d_qvert,int * d_dvert,int *d_delist,bool *d_qtree){
	int i;
	//found[0]=false;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	int n,p,j,k,flag=0;
	//for(ver=0;ver<d_qverc[0];ver++){
		int l=d_qvert[ver+1];
		int dver=qdmap[ver];
		
		n=d_dvert[dver+1];
		for(i=d_qvert[ver];i<l;i++)
		{
			flag=0;
			j=d_qelist[i];
			//if(!d_qtree[i])
			//	continue;
			p=d_dvert[dver];
			k=qdmap[j];
			for(;p<n;p++){
				if(k==d_delist[p]){
					flag=1;
					break;
				}
			}
			if(!flag){
				//*found=false;
				if(d_qtree[i]){				
					found[0]=found[1]=-1;		
					return;
				}
				else
					found[1]=0,found[0]++;
			}
		}
	//}
}

int * qdmap;
int *d_qverc,*d_dverc;
int *d_qvid,*d_qidtov,*h_qidtov,*h_qvid;
int *d_qvert,*d_qelist,*d_dvert,*d_delist;//,*d_dvelist,*d_qvelist;
bool *d_qtree,*d_over;	
int *d_qdmap;
bool h_over;
bool *h_qtree;
int *d_size_cvs,*h_size_cvs,ansi=0,treeansi=0;
long long int * h_anslist,*d_anslist;
long long int * h_treeanslist,*d_treeanslist;
int *h_treeremlist,*d_treeremlist;
__global__ void processoperation(int type,int a,int b,int nans,long long int *anslist,int *qverc,int *dverc,int *qvert,int *qelist,int *dvert,int *delist,int **cvsverlist,int *size_cvs,int *qvid){
	int ansi=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ansi>=nans)
		return ;
	long long int indexperm=anslist[ansi];
	if(indexperm==-1)
		return;
	//int *d_qdmap=new int[d_qverc[0]];//&d_mapans[d_qverc[0]*threadId];//new int[d_qverc[0]];
        int mapvera,mapverb=-1;
	int *aedges=NULL,till,i;
	//printf("%d\n ",indexperm);
	//anslist[ansi]=-1;
	for(i=0;i<qverc[0];i++){
                int j=qvid[i];
//                printf("j%d %d %d %dj ",i,j,size_cvs[j],mapvera);
		if(type==0 && i==a)
			mapvera=cvsverlist[j][indexperm%size_cvs[j]],aedges=&delist[dvert[mapvera]],till=dvert[mapvera+1];
                else if(type==0 && i==b)
			mapverb=cvsverlist[j][indexperm%size_cvs[j]];
		else if(type==1 && cvsverlist[j][indexperm%size_cvs[j]]==a)
			mapvera=i,aedges=&qelist[qvert[mapvera]],till=qvert[mapvera+1];
		else if(type==1 && cvsverlist[j][indexperm%size_cvs[j]]==b)
			mapverb=i;
                indexperm/=size_cvs[j];
        }
	bool flag=false;
//		anslist[ansi]=-1;
	if(aedges==NULL || mapverb==-1 || indexperm>0)
		return;
//	printf("j%d %dj",aedges[0],mapverb);
	
	for(i=0;i<till;i++){
		if(aedges[i]==mapverb){
			if(type==1){
				anslist[ansi]=-1;
				break;
			}
			flag=true;
			break;
		}
	}
	if(!flag && type==0)
		anslist[ansi]=-1;
}

__global__ void processqdnontree(int type,int a,int b,int ntans,long long int *tanslist,int *tremlist,int *qverc,int *dverc,int *qvert,int *qelist,int *dvert,int *delist,int **cvsverlist,int *size_cvs,int *qvid){
	int ansi=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ansi>=ntans)
		return ;
	long long int indexperm=tanslist[ansi];
	if(tremlist[ansi]==0)
		return;
        int mapvera,mapverb=-1;
	int *aedges=NULL,till,i;
	for(i=0;i<qverc[0];i++){
                int j=qvid[i];
		if(i==a)
			mapvera=cvsverlist[j][indexperm%size_cvs[j]],aedges=&delist[dvert[mapvera]],till=dvert[mapvera+1];
		else if(i==b)
			mapverb=cvsverlist[j][indexperm%size_cvs[j]];
                indexperm/=size_cvs[j];
        }
	
	bool flag=false;
//		anslist[ansi]=-1;
	if(aedges==NULL || mapverb==-1 || indexperm>0)
		return;
//	printf("j%d %dj",aedges[0],mapverb);
	for(i=0;i<till;i++){
		if(aedges[i]==mapverb){
			flag=true;
			break;
		}
	}
	if(!flag)
		 atomicDec((unsigned int *)&tremlist[ansi],tremlist[ansi]);
}

__global__ void doquery(int *nquery,int * type,int * vera,int * verb,int *ntans,long long int * tanslist,int *tremlist,int *nans,long long int * anslist,int ** cvsmatrix,int **cvslist,int *qverc,int *dverc,int *qvert,int *qelist,int *dvert,int *delist,int *size_cvs,int *qvid,bool *qtree){
	
	int qi=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(qi>=nquery[0])
		return ;
	//nans[0]=1;
	dim3 blocks((sqrtf(nans[0])/16 )+ 1,(sqrtf(nans[0])/16)+1);
	dim3 threads(16,16);
	if(type[qi]==0){
		processoperation<<<blocks,threads>>>(type[qi],vera[qi],verb[qi],nans[0],anslist,qverc,dverc,qvert,qelist,dvert,delist,cvslist,size_cvs,qvid);
	}
	else if(type[qi]==1){
		processoperation<<<blocks,threads>>>(type[qi],vera[qi],verb[qi],nans[0],anslist,qverc,dverc,qvert,qelist,dvert,delist,cvslist,size_cvs,qvid);
	}
	else if(type[qi]==3){
		int l=qvert[vera[qi]+1],i;
		bool flag=false,istree=false;
		for(i=qvert[vera[qi]];i<l;i++){
			if(qelist[i]==verb[qi]){
				flag=true;
				if(qtree[i])
					istree=true;
				break;
			}
		}
		if(!flag)
			return;
		if(!istree){
			processqdnontree<<<blocks,threads>>>(type[qi],vera[qi],verb[qi],ntans[0],tanslist,tremlist,qverc,dverc,qvert,qelist,dvert,delist,cvslist,size_cvs,qvid);
		}
		else{
			//processqdtree<<<blocks,threads>>>(type[qi],vera[qi],verb[qi],ntans[0],tanslist,tremlist,qverc,dverc,qvert,qelist,dvert,delist,cvslist,size_cvs,qvid);
		}
	}
	else{
	}

}

void callforallperm(bool * check,int ** cvslist,int i,int max,int dmax,long long int mapid){
	int j,k,l;
	l=h_qvid[i-1];
	//printf("i%d %di",i,l);
	if(i==max){
		for(j=0;j<dmax;j++)
			if(cvslist[l][j] && !check[j]){
				qdmap[i-1]=j;
				//mapid+=j*h_size_cvs[l];
				dim3 blocks((sqrt(max)/16 )+ 1,(sqrt(max)/16)+1);
				dim3 threads(16,16);
				int h_found[2]={0,1};
				int *d_found;
				hipMalloc(&d_found,sizeof(int)*2);
				hipMemcpy(d_found, h_found, sizeof(int)*2, hipMemcpyHostToDevice) ;
				hipMemcpy(d_qdmap, qdmap, sizeof(int)*(max+1), hipMemcpyHostToDevice);
				checkperm<<<blocks,threads>>> (d_found,d_qdmap,d_qverc,d_qelist,d_qvert,d_dvert,d_delist,d_qtree);
				//checkperm(bool *found,int * qdmap,int * d_qverc,int * d_qelist,int * d_qvert,int * d_dvert,int *d_delist)
				hipError_t err = hipGetLastError();
					if(err!=hipSuccess)
					{
						printf("Error: %s\n", hipGetErrorString(err));
						printf("Not Ok");
					}
				hipMemcpy(h_found, d_found, sizeof(int)*2, hipMemcpyDeviceToHost) ;
				if(h_found[1]==1){
					//for(k=0;k<max;k++)
					//	printf("%d ",qdmap[k]);
					//printf("\n");
					//	printf("OK\n");
					h_anslist[ansi++]=mapid*h_size_cvs[l]+j;
				}
				else if(h_found[0]!=-1){
					h_treeanslist[treeansi++]=mapid*h_size_cvs[l]+j;
					h_treeremlist[treeansi++]=h_found[0]/2;
				}
				//printf("\n");
			}
	}
	else{
		for(j=0;j<dmax;j++){
				//printf("%d %d %d\n",j,check[j],cvslist[l][j]);
				if(cvslist[l][j] && !check[j]){
				//ansi++;
				check[j]=true;
				qdmap[i-1]=j;
				//mapid+=l*h_size_cvs[l];
				callforallperm(check,cvslist,i+1,max,dmax,mapid*h_size_cvs[l] +j );
				check[j]=false;
			}
		}
	}
}

int main(int argc, char **argv)
{
	int deviceId = 4;
	hipSetDevice(deviceId);
	int h_qverc,h_dverc;
	
	
	int *h_qvert,*h_qelist,*h_dvert,*h_delist;//,*h_dvelist,*h_qvelist;
	
	int *d_hash1,*d_hash2;
	
	int i,j;
	int **h_cvslist,**d_cvslist,**h_tem;
	scanf("%d",&h_qverc);
	h_qvert=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_qvid=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_qidtov=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_tem=(int **)malloc(sizeof(int*)*(h_qverc+1));
	h_cvslist=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		scanf("%d",&h_qvert[i]);
	}
	h_qelist=(int *)malloc(sizeof(int)*h_qvert[h_qverc]);
	for(i=0;i<h_qvert[h_qverc];i++)
		scanf("%d",&h_qelist[i]);
	h_qtree=(bool *)malloc(sizeof(bool)*h_qvert[h_qverc]);
	for(i=0;i<h_qvert[h_qverc];i++){
		scanf("%d",&j);
		if(j==1)
			h_qtree[i]=true;
		else
			h_qtree[i]=false;
	}
	
	scanf("%d",&h_dverc);
	h_dvert=(int *)malloc(sizeof(int)*(h_dverc+1));
	for(i=0;i<=h_dverc;i++){
		scanf("%d",&h_dvert[i]);
	}
	for(i=0;i<=h_qverc;i++)
		h_cvslist[i]=(int *)malloc(sizeof(int)*(h_dverc+1));
	
	h_delist=(int *)malloc(sizeof(int)*h_dvert[h_dverc]);
	for(i=0;i<h_dvert[h_dverc];i++)
		scanf("%d",&h_delist[i]);
	hipMalloc(&d_qverc,sizeof(int));
	hipMalloc(&d_over,sizeof(bool));
	hipMalloc(&d_qvert,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qidtov,sizeof(int)*(h_qverc+1));
	//hipMalloc(&d_loc,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qelist,sizeof(int)*h_qvert[h_qverc]);
	hipMalloc(&d_qtree,sizeof(bool)*h_qvert[h_qverc]);
	hipMalloc(&d_hash1,sizeof(int)*1000038);
	hipMalloc(&d_hash2,sizeof(int)*1000038);
	
	hipMalloc(&d_qvid,sizeof(int)*(h_qverc+1));
	hipMemcpy(d_qverc,&h_qverc,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_qvert,h_qvert,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_qelist,h_qelist,sizeof(int)*h_qvert[h_qverc],hipMemcpyHostToDevice);
	hipMemcpy(d_qtree,h_qtree,sizeof(bool)*h_qvert[h_qverc],hipMemcpyHostToDevice);
	hipMemset(d_hash1,0,sizeof(int)*1000038);
	hipMemset(d_hash2,0,sizeof(int)*1000038);
	//hipMemset(d_loc,0,sizeof(int)*(h_qverc+1));
	hipMemset(d_qidtov,-1,sizeof(int)*(h_qverc+1));
	hipMemset(d_qvid,0,sizeof(int)*(h_qverc+1));
	int *h_hash1=(int *)malloc(sizeof(int)*1000038);
	int *h_hash2=(int *)malloc(sizeof(int)*1000038);
	dim3 blocks((sqrt(h_qverc)/16 )+ 1,(sqrt(h_qverc)/16)+1);
	dim3 threads(16,16);
	
	//int *d_qvert,int *d_dverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_hash1,bool *d_hash2)
	h_over=true;
	//h_qvid[1]=1;
	//h_qvid[3]=1;
	//hipMemcpy(d_qvid,h_qvid,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
//printf("qt%d %dqt\n",h_qtree[0],h_qtree[1]);
	setdeg1<<<blocks,threads>>>(d_qvert,d_qverc,d_qvid,d_qtree);

	h_over=false;
	int maxval=2;
	while(!h_over)
	{
		h_over=true;
		hipMemcpy(d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice) ;
		hipMemset(d_hash1,0,sizeof(int)*1000038);
		findhash <<<blocks,threads>>> (d_qvert,d_qverc,d_qvid,d_qelist,d_over,d_qtree,d_hash1,d_hash2);
		//(int *d_qvert,int *d_dverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_hash1,bool *d_qtree,bool *d_hash2)
		hipError_t err = hipGetLastError();
		if(err!=hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(err));
			printf("Not Ok");
		}
		hipMemcpy(h_hash1,d_hash1,sizeof(int)*1000038,hipMemcpyDeviceToHost);
		h_hash1[0]+=maxval;
		thrust::exclusive_scan(h_hash1,h_hash1+1000038,h_hash1);
		maxval=h_hash1[1000037];
		hipMemcpy(d_hash1,h_hash1,sizeof(int)*1000038,hipMemcpyHostToDevice);
		puttoid<<<blocks,threads>>>(d_qvert,d_qverc,d_qvid,d_qelist,d_qtree,d_hash1,d_qidtov);
///		hipMemcpy(h_hash2,d_hash2,sizeof(bool)*1000038,hipMemcpyDeviceToHost);
		hipMemcpy(&h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
		
		//printf("over flag:%d ",h_over);
		/*for(i=0;i<h_qverc;i++){
			//if()
				printf("%d ",h_qvid[i]);
//			if(h_hash2[i])
//				printf("h2 %d ",i);
//			if(h_hash1[i] || h_hash2[i])
//				printf("\n");
		}
		printf("\n");*/

	}	
	hipMemcpy(h_qvid,d_qvid,sizeof(int)*h_qverc,hipMemcpyDeviceToHost);
	hipMemcpy(h_qidtov,d_qidtov,sizeof(int)*(h_qverc+1),hipMemcpyDeviceToHost);
	for(i=0;i<=h_qverc;i++){
		printf("%d ",h_qidtov[i]);
	}
	printf("\n");
	for(i=0;i<=h_qverc;i++){
		printf("%d ",h_qvid[i]);
	}
	printf("\n");
	hipFree(d_qtree);
	hipFree(d_hash1);
	hipFree(d_hash2);
	free(h_hash1);
	free(h_hash2);
	free(h_qtree);
	
	hipMalloc(&d_cvslist,sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&h_tem[i],sizeof(int)*(h_dverc+1));
		hipMemset(h_tem[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemset(h_tem[1],1,sizeof(int)*(h_dverc+1));
	hipMemcpy(d_cvslist,h_tem,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMalloc(&d_dvert,sizeof(int)*(h_dverc+1));
	hipMalloc(&d_dverc,sizeof(int));
	hipMalloc(&d_delist,sizeof(int)*h_dvert[h_dverc]);	
	hipMemcpy(d_dverc,&h_dverc,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_dvert,h_dvert,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_delist,h_delist,sizeof(int)*h_dvert[h_dverc],hipMemcpyHostToDevice);
	dim3 dblocks((sqrt(h_dverc)/16 )+ 1,(sqrt(h_dverc)/16)+1);
	dim3 dthreads(16,16);
	int **d_cvsverlist,**d_temverlist;
	memset(h_cvslist[1],1,sizeof(int)*(h_dverc+1));
	h_size_cvs=(int *)malloc(sizeof(int)*(h_qverc+1));
	memset(h_size_cvs,0,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_size_cvs,sizeof(int)*(h_qverc+1));
	hipMemset(d_size_cvs,0,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_cvsverlist,sizeof(int*)*(h_qverc+1));
	d_temverlist=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&d_temverlist[i],sizeof(int)*(h_dverc+1));
		hipMemset(d_temverlist[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemcpy(d_cvsverlist,d_temverlist,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);
	long long int totalthreads=1;
	int *h_temploc;
	h_temploc=(int *)malloc(sizeof(int)*(h_dverc+1));
	for(i=0;i<h_dverc;i++)
		h_temploc[i]=i;
	hipMemcpy(d_temverlist[1],h_temploc,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
	h_size_cvs[1]=h_dverc;
	bool *d_tempcheck;	
	hipMalloc(&d_tempcheck,sizeof(bool)*(h_dvert[h_dverc]+1));	
	printf("Starting cvs find\n");
	for(i=0;i<=h_qverc;i++)
	{
		if(h_qidtov[i]!=-1)
		{
		hipMemset(d_tempcheck,false,sizeof(bool)*h_dvert[h_dverc]);	

		//findcvs(int ver,int *d_dvert,int *d_dverc,int *d_delist,int *d_qvert,int *d_qelist,int *d_qvid,bool ** d_dcvslist )
		findcvs<<<dblocks,dthreads>>>(d_tempcheck,h_qidtov[i],d_dvert,d_dverc,d_delist,d_qvert,d_qelist,d_qvid,d_cvslist);
		hipError_t err = hipGetLastError();
	
		printf("id %d \n",i);
		hipMemcpy(h_cvslist[i],h_tem[i],sizeof(int)*(h_dverc+1),hipMemcpyDeviceToHost);
		//printf("%d ",h_qidtov[i]);
		thrust::exclusive_scan(h_cvslist[i],h_cvslist[i]+h_dverc+1,h_temploc);
		h_size_cvs[i]=h_temploc[h_dverc];
		hipMemcpy(h_tem[i],h_temploc,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
		puttolist<<<dblocks,dthreads>>>(d_dverc,h_tem[i],d_temverlist[i]);
		//for(j=0;j<=h_dverc;j++)
		//	if(h_cvslist[i][j])
		//		printf("%d ",j);
		//printf("\n");
	//	hipMemcpy(h_cvslist[i],d_temverlist[i],sizeof(int)*(h_dverc+1),hipMemcpyDeviceToHost);
		//hipMemcpy(h_temploc,h_tem[i],sizeof(int)*(h_size_cvs[i]),hipMemcpyDeviceToHost);
		//printf("On list");
		//for(j=0;j<h_size_cvs[i];j++)
		//	printf("%d ",h_temploc[j]);
		//printf("\n");	
		printf("size %d\n",h_size_cvs[i]);
		}	
		
	}
	hipMemcpy(d_size_cvs,h_size_cvs,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
	//hipMemcpy(h_delist,d_delist,sizeof(int)*(h_dvert[h_dverc]),hipMemcpyDeviceToHost);
	//	for(j=0;j<h_dvert[h_dverc];j++)
	//		printf("%d ",h_delist[j]);	
	hipFree(d_tempcheck);
	free(h_temploc);
	bool * check=(bool *)malloc(sizeof(bool)*(h_dverc+1));
	memset(check,false,sizeof(bool)*(h_dverc+1));
	qdmap=(int *)malloc(sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qdmap,sizeof(int)*(h_qverc+1));
	h_anslist=(long long int *)malloc(sizeof(long long int)*1000001);
	hipMalloc(&d_anslist,sizeof(long long int)*(1000001));
	h_treeanslist=(long long int*)malloc(sizeof(long long int)*1000001);
	hipMalloc(&d_treeanslist,sizeof(long long int)*(1000001));
	h_treeremlist=(int *)malloc(sizeof(int)*1000001);
	hipMalloc(&d_treeremlist,sizeof(int)*(1000001));
	ansi=0;
	callforallperm(check,h_cvslist,1,h_qverc,h_dverc,0);
	
	printf("Final:%d\n",ansi);
	//answers found
	int * d_ansi,*d_treeansi;
	hipMalloc(&d_ansi,sizeof(int));
	hipMemcpy(d_ansi,&ansi,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_anslist,h_anslist,sizeof(long long int)*(ansi),hipMemcpyHostToDevice);
	hipMalloc(&d_treeansi,sizeof(int));
	hipMemcpy(d_treeansi,&treeansi,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_treeanslist,h_treeanslist,sizeof(long long int)*(treeansi),hipMemcpyHostToDevice);
	hipMemcpy(d_treeremlist,h_treeremlist,sizeof(int)*(treeansi),hipMemcpyHostToDevice);
	

/*	int nqueries,*d_nqueries;
	int *h_vera,*h_verb;
	int *d_vera,*d_verb;
	int *h_type,*d_type;
	scanf("%d",&nqueries);	
	h_vera=(int*) malloc(sizeof(int)*nqueries);
	h_verb=(int*) malloc(sizeof(int)*nqueries);
	h_type=(int *) malloc(sizeof(int)*nqueries);
	
		
	hipMalloc(&d_vera,sizeof(int)*nqueries);
	hipMalloc(&d_verb,sizeof(int)*nqueries);
	hipMalloc(&d_type,sizeof(int)*nqueries);

	for(i=0;i<nqueries;i++){
		scanf("%d%d%d",&h_type[i],&h_vera[i],&h_verb[i]);
	}
	hipMemcpy(d_vera,h_vera,sizeof(int)*nqueries,hipMemcpyHostToDevice);
	hipMemcpy(d_verb,h_verb,sizeof(int)*nqueries,hipMemcpyHostToDevice);
	hipMemcpy(d_type,h_type,sizeof(int)*nqueries,hipMemcpyHostToDevice);
	
	hipMalloc(&d_nqueries,sizeof(int));
	hipMemcpy(d_nqueries,&nqueries,sizeof(int),hipMemcpyHostToDevice);
	
	dim3 qblocks((sqrt(nqueries)/16 )+ 1,(sqrt(nqueries)/16)+1);
	dim3 qthreads(16,16);

	doquery<<<qblocks,qthreads>>>(d_nqueries,d_type,d_vera,d_verb,d_treeansi,d_treeanslist,d_treeremlist,d_ansi,d_anslist,d_cvslist,d_cvsverlist,d_qverc,d_dverc,d_qvert,d_qelist,d_dvert,d_delist,d_size_cvs,d_qvid,d_qtree);

	
	hipMemcpy(h_anslist,d_anslist,sizeof(long long int)*(ansi),hipMemcpyDeviceToHost);
	for(i=0;i<ansi;i++)
		if(h_anslist[i]==-1)
			printf(" %d ",i);
*/	hipFree(d_over);
	hipFree(d_qverc);
	hipFree(d_qvert);
	hipFree(d_qelist);
	hipFree(d_qvid);
	hipFree(d_qidtov);
	hipFree(d_dvert);
	hipFree(d_delist);
	hipFree(d_dverc);
	hipFree(d_cvslist);
	hipFree(d_cvsverlist);
	hipFree(d_size_cvs);
	hipFree(d_anslist);
	/*free(h_qvid);
	free(h_qvert);
	//free(h_qelist);
	free(h_qidtov);
	free(h_cvslist);
	free(h_dvert);
	free(h_delist);*/
}	
