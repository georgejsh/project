#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include ""
#include <thrust/scan.h>
const int BASE1 = 10000 + 7;
const int BASE2 = 100000 + 3;
const int MOD1 = 1000000 + 3;
const int MOD2 = 1000000 + 37;
__global__ void findhash(int *d_qvert,int *d_qverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_qtree,int *d_hash1,int *d_hash2)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	int l=d_qvert[ver+1];
	int hash1=1,hash2=1;
	int flag=0;
	for(i=d_qvert[ver];i<l;i++)
	{
		int m=d_qelist[i];
		bool treeedge=d_qtree[i];
		if(treeedge){
			int tt=d_qvid[m];
			if(tt==0)
				return;
			flag=1;
			hash1=(hash1*1L*BASE1)*tt % MOD1;
			hash2=(hash2*1L*BASE2)*tt % MOD2;
		}
	}
	if(flag==0)
		return;
	if(flag==1){
		*d_over=false;
		d_hash1[hash1]=1;
		d_hash2[hash2]=1;
	}
	
}
__global__ void setdeg1(int *d_qvert,int *d_qverc,int *d_qvid,bool *d_qtree)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	
	int l=d_qvert[ver+1];
	bool treeedge;
	
	for(i=d_qvert[ver];i<l;i++)
	{
		treeedge=d_qtree[i];
		
		if(treeedge)
			return;
	}
//printf("%d %d\n",ver,i);
	d_qvid[ver]=1;	
}
/*__global__ void alignhash(bool *d_hash1,bool *d_hash2)
{
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=1000038)
		return ;
	if(d_hash1[ver] || d_hash2[ver]){
		d_hash1=true;
	}
}*/
__global__ void puttoid(int *d_qvert,int *d_qverc,int *d_qvid,int *d_qelist,bool *d_qtree,int *d_loc,int * d_qidtov)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	int l=d_qvert[ver+1];
	int hash1=1,hash2=1;
	int flag=0;
	for(i=d_qvert[ver];i<l;i++)
	{
		int m=d_qelist[i];
		bool treeedge=d_qtree[i];
		if(treeedge){
			int tt=d_qvid[m];
			if(tt==0)
				return;
			flag=1;
			hash1=(hash1*1L*BASE1)*tt % MOD1;
			hash2=(hash2*1L*BASE2)*tt % MOD2;
		}
	}
	//printf("%d %d %d \n",ver,flag,d_loc[hash1]);
	if(flag==0)
		return;
	int id=d_loc[hash1];
	d_qvid[ver]=id;
	d_qidtov[id]=ver;
}
__device__ bool chechall(int ver,bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,int ** d_dcvslist){
	//int ql=qfrom-qto;
	int ql=qto-qfrom;
	int j,k,l;
	//d_dcvslist[2][ql]=true;
	if(i==ql){
		
		k=d_qelist[i+qfrom-1];
		k=d_qvid[k];
		if(k>=d_qvid[ver])
			return true;
		for(j=dfrom;j<dto;j++){
			l=d_delist[j];
			if(check[l])
				continue;
			if(!d_dcvslist[k][l])
				continue;
			return true;
		}
	}
	else{
		int res=false;
		k=d_qelist[i+qfrom-1];
		k=d_qvid[k];
		if(k>=d_qvid[ver])
			return chechall(ver,check,i+1,dfrom,dto,d_delist,d_qelist,d_qvid,qfrom,qto,d_dcvslist);
		
		for(j=dfrom;j<dto;j++){
			l=d_delist[j];
			if(check[l])
				continue;
			if(!d_dcvslist[k][l])
				continue;
			check[l]=true;
			res|=chechall(ver,check,i+1,dfrom,dto,d_delist,d_qelist,d_qvid,qfrom,qto,d_dcvslist);
			if(res==true)
				return true;
			check[l]=false;
		}
	}
	return false;
}
__global__ void findcvs(int ver,int *d_dvert,int *d_dverc,int *d_delist,int *d_qvert,int *d_qelist,int *d_qvid,int ** d_dcvslist )
{
	//int i;
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	int ql=d_qvert[ver+1]-d_qvert[ver];
	int dl=d_dvert[dver+1]-d_dvert[dver];
	if(ql>dl)
		return;
	bool *checked=(bool*)malloc(sizeof(bool)*d_dverc[0]);
	//bool *checked=new bool[d_dverc[0]];
	memset(checked,false,sizeof(bool)*d_dverc[0]);
	//chechall(bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,bool ** d_dcvslist)
	if(chechall(ver,checked,1,d_dvert[dver],d_dvert[dver+1],d_delist,d_qelist,d_qvid,d_qvert[ver],d_qvert[ver+1],d_dcvslist))
		d_dcvslist[d_qvid[ver]][dver]=true;
	free(checked);
}
__global__ void puttolist(int *d_dverc,int *d_loc,int * d_dcvslist )
{
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	if(d_loc[dver]!=d_loc[dver+1])
		d_dcvslist[d_loc[dver]]=dver;
}
__global__ void checkperm(bool *found,int * qdmap,int * d_qverc,int * d_qelist,int * d_qvert,int * d_dvert,int *d_delist){
	int i;
	//found[0]=false;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	int n,p,j,k,flag=0;
	//for(ver=0;ver<d_qverc[0];ver++){
		int l=d_qvert[ver+1];
		int dver=qdmap[ver];
		
		n=d_dvert[dver+1];
		for(i=d_qvert[ver];i<l;i++)
		{
			flag=0;
			j=d_qelist[i];
			p=d_dvert[dver];
			k=qdmap[j];
			for(;p<n;p++){
				if(k==d_delist[p]){
					flag=1;
					break;
				}
			}
			if(!flag){
				*found=false;
				return;
			}
		}
	//}
}

__global__ void findall(int *d_mapans,int *d_cans,int *d_qvid,int * d_qverc,int * d_qelist,int * d_qvert,int * d_dvert,int *d_delist,int ** d_cvsverlist,int * d_size_cvs)
{
	bool found[1]={true};
	long long int blockId = blockIdx.x 
			 + blockIdx.y * gridDim.x 
			 + gridDim.x * gridDim.y * blockIdx.z; 
	long long int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
			  + (threadIdx.z * (blockDim.x * blockDim.y))
			  + (threadIdx.y * blockDim.x)
			  + threadIdx.x;
	int i=0;
	long long int indexperm=threadId;
	for(i=0;i<d_qverc[0];i++){
		int j=d_qvid[i];
		indexperm/=d_size_cvs[j];
	}
	if(indexperm)
		return;
	indexperm=threadId;	
	int *d_qdmap=&d_mapans[d_qverc[0]*threadId];
	for(i=0;i<d_qverc[0];i++){
		int j=d_qvid[i];
		d_qdmap[i]=d_cvsverlist[j][indexperm%d_size_cvs[j]];
		indexperm/=d_size_cvs[j];
	}
	
	//dim3 blocks((max/16 )+ 1,(max/16)+1);
	//dim3 threads(16,16);
	//found[0]=true;
	//checkperm<<<blocks,threads>>> (found,d_qdmap,d_qverc,d_qelist,d_qvert,d_dvert,d_delist);
	int n,p,j,k,flag=0,ver;
	for(ver=0;ver<d_qverc[0];ver++){
		int l=d_qvert[ver+1];
		int dver=d_qdmap[ver];
		
		n=d_dvert[dver+1];
		for(i=d_qvert[ver];i<l;i++)
		{
			flag=0;
			j=d_qelist[i];
			p=d_dvert[dver];
			k=d_qdmap[j];
			for(;p<n;p++){
				if(k==d_delist[p]){
					flag=1;
					break;
				}
			}
			if(!flag){
				*found=false;
				return;
			}
		}
	}
	if(found[0]){
		//printf("%d ", threadId);
	}
}
int * qdmap;
int *d_qverc,*d_dverc;
int *d_qvid,*d_qidtov,*h_qidtov,*h_qvid;
int *d_qvert,*d_qelist,*d_dvert,*d_delist;//,*d_dvelist,*d_qvelist;
bool *d_qtree,*d_over;	
int *d_qdmap;
bool h_over;
/*void callforallperm(bool * check,int ** cvslist,int i,int max,int dmax){
	int j,k,l;
	l=h_qvid[i-1];
	//printf("i%d %di",i,l);
	if(i==max){
		for(j=0;j<dmax;j++)
			if(cvslist[l][j] && !check[j]){
				qdmap[i-1]=j;
				dim3 blocks((max/16 )+ 1,(max/16)+1);
				dim3 threads(16,16);
				h_over=true;
				//for(k=0;k<max;k++)
				//		printf("%d ",qdmap[k]);
				
				hipMemcpy(d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice) ;
				hipMemcpy(d_qdmap, qdmap, sizeof(int)*(max+1), hipMemcpyHostToDevice);
				checkperm<<<blocks,threads>>> (d_over,d_qdmap,d_qverc,d_qelist,d_qvert,d_dvert,d_delist);
				//checkperm(bool *found,int * qdmap,int * d_qverc,int * d_qelist,int * d_qvert,int * d_dvert,int *d_delist)
				hipError_t err = hipGetLastError();
					if(err!=hipSuccess)
					{
						printf("Error: %s\n", hipGetErrorString(err));
						printf("Not Ok");
					}
				hipMemcpy(&h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
				if(h_over){
					for(k=0;k<max;k++)
						printf("%d ",qdmap[k]);
					//printf("\n");
						printf("OK\n");
				}
				//printf("\n");
			}
	}
	else{
		for(j=0;j<dmax;j++){
				//printf("%d %d %d\n",j,check[j],cvslist[l][j]);
				if(cvslist[l][j] && !check[j]){
				check[j]=true;
				qdmap[i-1]=j;
				callforallperm(check,cvslist,i+1,max,dmax);
				check[j]=false;
			}
		}
	}
}*/

int main(int argc, char **argv)
{
	int deviceId = 4;
	hipSetDevice(deviceId);
	int h_qverc,h_dverc;
	
	
	int *h_qvert,*h_qelist,*h_dvert,*h_delist;//,*h_dvelist,*h_qvelist;
	
	bool *h_qtree;
	int *d_hash1,*d_hash2;
	
	int i,j;
	int **h_cvslist,**d_cvslist,**h_tem;
	scanf("%d",&h_qverc);
	h_qvert=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_qvid=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_qidtov=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_tem=(int **)malloc(sizeof(int*)*(h_qverc+1));
	h_cvslist=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		scanf("%d",&h_qvert[i]);
	}
	h_qelist=(int *)malloc(sizeof(int)*h_qvert[h_qverc]);
	for(i=0;i<h_qvert[h_qverc];i++)
		scanf("%d",&h_qelist[i]);
	h_qtree=(bool *)malloc(sizeof(bool)*h_qvert[h_qverc]);
	for(i=0;i<h_qvert[h_qverc];i++){
		scanf("%d",&j);
		if(j==1)
			h_qtree[i]=true;
		else
			h_qtree[i]=false;
	}
	
	scanf("%d",&h_dverc);
	h_dvert=(int *)malloc(sizeof(int)*(h_dverc+1));
	for(i=0;i<=h_dverc;i++){
		scanf("%d",&h_dvert[i]);
	}
	for(i=0;i<=h_qverc;i++)
		h_cvslist[i]=(int *)malloc(sizeof(int)*(h_dverc+1));
	
	h_delist=(int *)malloc(sizeof(int)*h_dvert[h_dverc]);
	for(i=0;i<h_dvert[h_dverc];i++)
		scanf("%d",&h_delist[i]);
	hipMalloc(&d_qverc,sizeof(int));
	hipMalloc(&d_over,sizeof(bool));
	hipMalloc(&d_qvert,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qidtov,sizeof(int)*(h_qverc+1));
	//hipMalloc(&d_loc,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qelist,sizeof(int)*h_qvert[h_qverc]);
	hipMalloc(&d_qtree,sizeof(bool)*h_qvert[h_qverc]);
	hipMalloc(&d_hash1,sizeof(int)*1000038);
	hipMalloc(&d_hash2,sizeof(int)*1000038);
	
	hipMalloc(&d_qvid,sizeof(int)*(h_qverc+1));
	hipMemcpy(d_qverc,&h_qverc,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_qvert,h_qvert,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_qelist,h_qelist,sizeof(int)*h_qvert[h_qverc],hipMemcpyHostToDevice);
	hipMemcpy(d_qtree,h_qtree,sizeof(bool)*h_qvert[h_qverc],hipMemcpyHostToDevice);
	hipMemset(d_hash1,0,sizeof(int)*1000038);
	hipMemset(d_hash2,0,sizeof(int)*1000038);
	//hipMemset(d_loc,0,sizeof(int)*(h_qverc+1));
	hipMemset(d_qidtov,-1,sizeof(int)*(h_qverc+1));
	hipMemset(d_qvid,0,sizeof(int)*(h_qverc+1));
	int *h_hash1=(int *)malloc(sizeof(int)*1000038);
	int *h_hash2=(int *)malloc(sizeof(int)*1000038);
	dim3 blocks((h_qverc/16 )+ 1,(h_qverc/16)+1);
	dim3 threads(16,16);
	
	//int *d_qvert,int *d_dverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_hash1,bool *d_hash2)
	h_over=true;
	//h_qvid[1]=1;
	//h_qvid[3]=1;
	//hipMemcpy(d_qvid,h_qvid,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
//printf("qt%d %dqt\n",h_qtree[0],h_qtree[1]);
	setdeg1<<<blocks,threads>>>(d_qvert,d_qverc,d_qvid,d_qtree);

	h_over=false;
	int maxval=2;
	while(!h_over)
	{
		h_over=true;
		hipMemcpy(d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice) ;
		hipMemset(d_hash1,0,sizeof(int)*1000038);
		findhash <<<blocks,threads>>> (d_qvert,d_qverc,d_qvid,d_qelist,d_over,d_qtree,d_hash1,d_hash2);
		//(int *d_qvert,int *d_dverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_hash1,bool *d_qtree,bool *d_hash2)
		hipError_t err = hipGetLastError();
		if(err!=hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(err));
			printf("Not Ok");
		}
		hipMemcpy(h_hash1,d_hash1,sizeof(int)*1000038,hipMemcpyDeviceToHost);
		h_hash1[0]+=maxval;
		thrust::exclusive_scan(h_hash1,h_hash1+1000038,h_hash1);
		maxval=h_hash1[1000037];
		hipMemcpy(d_hash1,h_hash1,sizeof(int)*1000038,hipMemcpyHostToDevice);
		puttoid<<<blocks,threads>>>(d_qvert,d_qverc,d_qvid,d_qelist,d_qtree,d_hash1,d_qidtov);
///		hipMemcpy(h_hash2,d_hash2,sizeof(bool)*1000038,hipMemcpyDeviceToHost);
		hipMemcpy(&h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
		
		//printf("over flag:%d ",h_over);
		/*for(i=0;i<h_qverc;i++){
			//if()
				printf("%d ",h_qvid[i]);
//			if(h_hash2[i])
//				printf("h2 %d ",i);
//			if(h_hash1[i] || h_hash2[i])
//				printf("\n");
		}
		printf("\n");*/

	}	
	hipMemcpy(h_qvid,d_qvid,sizeof(int)*h_qverc,hipMemcpyDeviceToHost);
	hipMemcpy(h_qidtov,d_qidtov,sizeof(int)*(h_qverc+1),hipMemcpyDeviceToHost);
	for(i=0;i<=h_qverc;i++){
		printf("%d ",h_qidtov[i]);
	}
	printf("\n");
	for(i=0;i<=h_qverc;i++){
		printf("%d ",h_qvid[i]);
	}
	printf("\n");
	hipFree(d_qtree);
	hipFree(d_hash1);
	hipFree(d_hash2);
	free(h_hash1);
	free(h_hash2);
	free(h_qtree);
	
	hipMalloc(&d_cvslist,sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&h_tem[i],sizeof(int)*(h_dverc+1));
		hipMemset(h_tem[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemset(h_tem[1],1,sizeof(int)*(h_dverc+1));
	hipMemcpy(d_cvslist,h_tem,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMalloc(&d_dvert,sizeof(int)*(h_dverc+1));
	hipMalloc(&d_dverc,sizeof(int));
	hipMalloc(&d_delist,sizeof(int)*h_dvert[h_dverc]);	
	hipMemcpy(d_dverc,&h_dverc,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_dvert,h_dvert,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_delist,h_delist,sizeof(int)*h_dvert[h_dverc],hipMemcpyHostToDevice);
	dim3 dblocks((h_dverc/16 )+ 1,(h_dverc/16)+1);
	dim3 dthreads(16,16);
	int **d_cvsverlist,**d_temverlist;
	int *d_size_cvs,*h_size_cvs;
	memset(h_cvslist[1],1,sizeof(int)*(h_dverc+1));
	h_size_cvs=(int *)malloc(sizeof(int)*(h_qverc+1));
	memset(h_size_cvs,0,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_size_cvs,sizeof(int)*(h_qverc+1));
	hipMemset(d_size_cvs,0,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_cvsverlist,sizeof(int*)*(h_qverc+1));
	d_temverlist=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&d_temverlist[i],sizeof(int)*(h_dverc+1));
		hipMemset(d_temverlist[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemcpy(d_cvsverlist,d_temverlist,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);
	long long int totalthreads=1;
	for(i=0;i<h_dverc;i++)
		h_cvslist[1][i]=i;
	hipMemcpy(d_temverlist[1],h_cvslist[1],sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
	h_size_cvs[1]=h_dverc;
	for(i=0;i<=h_qverc;i++)
	{
		if(h_qidtov[i]!=-1)
		{

		//findcvs(int ver,int *d_dvert,int *d_dverc,int *d_delist,int *d_qvert,int *d_qelist,int *d_qvid,bool ** d_dcvslist )
		findcvs<<<dblocks,dthreads>>>(h_qidtov[i],d_dvert,d_dverc,d_delist,d_qvert,d_qelist,d_qvid,d_cvslist);
		hipError_t err = hipGetLastError();
	
		hipMemcpy(h_cvslist[i],h_tem[i],sizeof(int)*(h_dverc+1),hipMemcpyDeviceToHost);
		for(j=0;j<=h_dverc;j++)
			if(h_cvslist[i][j])
				printf("%d ",j);
		printf("\n");
		//printf("%d ",h_qidtov[i]);
		thrust::exclusive_scan(h_cvslist[i],h_cvslist[i]+h_dverc+1,h_cvslist[i]);
		h_size_cvs[i]=h_cvslist[i][h_dverc];
		hipMemcpy(h_tem[i],h_cvslist[i],sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
		puttolist<<<dblocks,dthreads>>>(d_dverc,h_tem[i],d_temverlist[i]);
	//	hipMemcpy(h_cvslist[i],d_temverlist[i],sizeof(int)*(h_dverc+1),hipMemcpyDeviceToHost);
	//	for(j=0;j<=h_dverc;j++)
	//		printf("%d ",h_cvslist[i][j]);	
		}	
		
	}
	//	hipMemcpy(h_delist,d_delist,sizeof(int)*(h_dvert[h_dverc]),hipMemcpyDeviceToHost);
	//	for(j=0;j<h_dvert[h_dverc];j++)
	//		printf("%d ",h_delist[j]);	
			
	for(i=0;i<h_qverc;i++)
		if(h_size_cvs[h_qvid[i]])
		totalthreads*=h_size_cvs[h_qvid[i]];
	printf("Start %lld\n",totalthreads);
	hipMemcpy(d_size_cvs,h_size_cvs,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
	//totalthreads=1000;
	dim3 dpblocks(((int)(totalthreads/8 )+ 1),((int)(totalthreads/8)+1),((int)(totalthreads/8)+1));
	dim3 dpthreads(8,8,8);
	 int *d_mapans,*d_countans,*h_countans;
	hipMalloc(&d_mapans,sizeof( int)*totalthreads*(h_qverc+1));
	hipMalloc(&d_countans,sizeof( int)*(totalthreads+1));
	hipMemset(d_countans,0,sizeof( int)*(totalthreads+1));
	h_countans=(int *)malloc(sizeof(int)*(totalthreads+1));
	//h_countans=0;
	//hipMemcpy(d_countans, &h_countans, sizeof( int), hipMemcpyHostToDevice) ;
	//hipMemcpy(&h_countans, d_qverc, sizeof(int), hipMemcpyDeviceToHost) ;
	//printf("%d\n",h_countans);
	findall<<<dpblocks,dpthreads>>> (d_mapans,d_countans,d_qvid,d_qverc,d_qelist,d_qvert,d_dvert,d_delist,d_cvsverlist,d_size_cvs);
	hipMemcpy(h_countans, d_countans, sizeof(int)*(totalthreads+1), hipMemcpyDeviceToHost) ;
	thrust::exclusive_scan(h_countans,h_countans+totalthreads+1,h_countans);
	printf("%d\n",h_countans[totalthreads] );
	//printf("%d\n",h_countans);
	/*j=0;
	for(i=0;i<totalthreads;i++)
		if(h_countans[i])
			j++;
	printf("%d\n",j);
	*///bool * check=(bool *)malloc(sizeof(bool)*(h_dverc+1));
	//memset(check,false,sizeof(bool)*(h_dverc+1));
	//qdmap=(int *)malloc(sizeof(int)*(h_qverc+1));
	//hipMalloc(&d_qdmap,sizeof(int)*(h_qverc+1));
	//callforallperm(check,h_cvslist,1,h_qverc,h_dverc);

	hipFree(d_over);
	hipFree(d_qverc);
	hipFree(d_qvert);
	hipFree(d_qelist);
	hipFree(d_qvid);
	hipFree(d_qidtov);
	hipFree(d_dvert);
	hipFree(d_delist);
	hipFree(d_dverc);
	hipFree(d_cvslist);
	hipFree(d_cvsverlist);
	hipFree(d_size_cvs);
	/*free(h_qvid);
	free(h_qvert);
	//free(h_qelist);
	free(h_qidtov);
	free(h_cvslist);
	free(h_dvert);
	free(h_delist);*/
}
	
