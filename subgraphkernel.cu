#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include ""
#include <thrust/scan.h>
const int BASE1 = 10000 + 7;
const int BASE2 = 100000 + 3;
const int MOD1 = 1000000 + 3;
const int MOD2 = 1000000 + 37;
__global__ void findhash(int *d_qvert,char *d_qverlabel,int *d_qverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_qtree,int *d_hash1,int *d_hash2)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	int l=d_qvert[ver+1];
	int hash1=d_qverlabel[ver],hash2=1;
	int flag=0;
	for(i=d_qvert[ver];i<l;i++)
	{
		int m=d_qelist[i];
		bool treeedge=d_qtree[i];
		if(treeedge){
			int tt=d_qvid[m];
			if(tt==0)
				return;
			flag=1;
			hash1=(hash1*1L*BASE1)*tt % MOD1;
		//	hash2=(hash2*1L*BASE2)*tt % MOD2;
		}
	}
	//if(flag==0)
	//	return;
	//if(flag==1)
	{
		*d_over=false;
		d_hash1[hash1]=1;
		//d_hash2[hash2]=1;
	}
	
}
__global__ void setdeg1(int *d_qvert,int *d_qverc,int *d_qvid,bool *d_qtree)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	
	int l=d_qvert[ver+1];
	bool treeedge;
	
	for(i=d_qvert[ver];i<l;i++)
	{
		treeedge=d_qtree[i];
		
		if(treeedge)
			return;
	}
//printf("%d %d\n",ver,i);
	d_qvid[ver]=1;	
}
/*__global__ void alignhash(bool *d_hash1,bool *d_hash2)
{
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=1000038)
		return ;
	if(d_hash1[ver] || d_hash2[ver]){
		d_hash1=true;
	}
}*/
__global__ void puttoid(int *d_qvert,char *d_qverlabel,int *d_qverc,int *d_qvid,int *d_qelist,bool *d_qtree,int *d_loc,int * d_qidtov,int *qparent)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	if(d_qvid[ver]!=0)
		return;
	int l=d_qvert[ver+1];
	int hash1=d_qverlabel[ver],hash2=1;
	int flag=0;
	for(i=d_qvert[ver];i<l;i++)
	{
		int m=d_qelist[i];
		bool treeedge=d_qtree[i];
		if(treeedge){
			int tt=d_qvid[m];
			if(tt==0)
				return;
			flag=1;
			hash1=(hash1*1L*BASE1)*tt % MOD1;
		//	hash2=(hash2*1L*BASE2)*tt % MOD2;
		}
	}
	
	for(i=d_qvert[ver];i<l;i++){
		int m=d_qelist[i];
		bool treeedge=d_qtree[i];
		if(treeedge){
			qparent[m]=ver;
		}
	}
	//printf("%d %d %d \n",ver,flag,d_loc[hash1]);
//	if(flag==0)
//		return;
	int id=d_loc[hash1];
	d_qvid[ver]=id;
	d_qidtov[id]=ver;
}
__device__ bool chechall(int ver,bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,int ** d_dcvslist){
	//int ql=qfrom-qto;
	int ql=qto-qfrom;
	int j,k,l;
	//d_dcvslist[2][ql]=true;
	if(i==ql){
		
		k=d_qelist[i+qfrom-1];
		k=d_qvid[k];
		if(k>=d_qvid[ver])
			return true;
		for(j=dfrom;j<dto;j++){
			l=d_delist[j];
			if(check[j])
				continue;
			//if(ver==0)
			//	printf("a%da",l);
			if(!d_dcvslist[k][l])
				continue;
			return true;
		}
	}
	else{
		int res=false;
		k=d_qelist[i+qfrom-1];
		k=d_qvid[k];
		if(k>=d_qvid[ver])
			return chechall(ver,check,i+1,dfrom,dto,d_delist,d_qelist,d_qvid,qfrom,qto,d_dcvslist);
		for(j=dfrom;j<dto;j++){
			l=d_delist[j];
			if(check[j])
				continue;
			if(!d_dcvslist[k][l])
				continue;
			check[j]=true;
			res|=chechall(ver,check,i+1,dfrom,dto,d_delist,d_qelist,d_qvid,qfrom,qto,d_dcvslist);
			if(res==true)
				return true;
			check[j]=false;
		}
	}
	return false;
}
__global__ void findcvs(bool *temp,int ver,int *d_dvert,char * d_dverlabel,int *d_dverc,int *d_delist,int *d_qvert,char *d_qverlabel,int *d_qelist,int *d_qvid,int ** d_dcvslist )
{
	//int i;
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	if(d_dverlabel[dver]!=d_qverlabel[ver])
		return;
	int ql=d_qvert[ver+1]-d_qvert[ver];
	int dl=d_dvert[dver+1]-d_dvert[dver];
	if(ql>dl)
		return;
//	if(dver!=1  && ver==0)
//		return;
//	printf("%d\n",dver);
	//bool *checked=(bool*)malloc(sizeof(bool)*d_dverc[0]);
	//bool *checked=new bool[d_dverc[0]];
	//memset(checked,false,sizeof(bool)*d_dverc[0]);
	//chechall(bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,bool ** d_dcvslist)
	if(chechall(ver,temp,1,d_dvert[dver],d_dvert[dver+1],d_delist,d_qelist,d_qvid,d_qvert[ver],d_qvert[ver+1],d_dcvslist))
		d_dcvslist[d_qvid[ver]][dver]=true;
	//free(checked);
}
__global__ void puttolist(int *d_dverc,int *d_loc,int * d_dcvslist )
{
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	if(d_loc[dver]!=d_loc[dver+1])
		d_dcvslist[d_loc[dver]]=dver;
}
__global__ void checkperm(int *found,int * qdmap,int * d_qverc,int * d_qelist,int * d_qvert,int * d_dvert,int *d_delist,bool *d_qtree){
	int i;
	//found[0]=false;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=d_qverc[0])
		return ;
	int n,p,j,k,flag=0;
	//for(ver=0;ver<d_qverc[0];ver++){
		int l=d_qvert[ver+1];
		int dver=qdmap[ver];
		
		n=d_dvert[dver+1];
		for(i=d_qvert[ver];i<l;i++)
		{
			flag=0;
			j=d_qelist[i];
			//if(!d_qtree[i])
			//	continue;
			p=d_dvert[dver];
			k=qdmap[j];
			for(;p<n;p++){
				if(k==d_delist[p]){
					flag=1;
					break;
				}
			}
			if(!flag){
				//*found=false;
				if(d_qtree[i]){				
					found[0]=found[1]=-1;		
					return;
				}
				else
					found[1]=0,found[0]++;
			}
		}
	//}
}

int * qdmap;
int *d_qverc,*d_dverc;
int *d_qvid,*d_qidtov,*h_qidtov,*h_qvid;
int *d_qvert,*d_qelist,*d_dvert,*d_delist;//,*d_dvelist,*d_qvelist;
bool *d_qtree,*d_over;	
int *d_qdmap;
bool h_over;
bool *h_qtree;
int *d_size_cvs,*h_size_cvs,ansi=0,treeansi=0;
long long int * h_anslist,*d_anslist;
long long int * h_treeanslist,*d_treeanslist;
int *h_treeremlist,*d_treeremlist;
/*__global__ void processoperation(int type,int a,int b,int nans,long long int *anslist,int *qverc,int *dverc,int *qvert,int *qelist,int *dvert,int *delist,int **cvsverlist,int *size_cvs,int *qvid){
	int ansi=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ansi>=nans)
		return ;
	long long int indexperm=anslist[ansi];
	if(indexperm==-1)
		return;
	//int *d_qdmap=new int[d_qverc[0]];//&d_mapans[d_qverc[0]*threadId];//new int[d_qverc[0]];
        int mapvera,mapverb=-1;
	int *aedges=NULL,till,i;
	//printf("%d\n ",indexperm);
	//anslist[ansi]=-1;
	for(i=0;i<qverc[0];i++){
                int j=qvid[i];
//                printf("j%d %d %d %dj ",i,j,size_cvs[j],mapvera);
		if(type==0 && i==a)
			mapvera=cvsverlist[j][indexperm%size_cvs[j]],aedges=&delist[dvert[mapvera]],till=dvert[mapvera+1];
                else if(type==0 && i==b)
			mapverb=cvsverlist[j][indexperm%size_cvs[j]];
		else if(type==1 && cvsverlist[j][indexperm%size_cvs[j]]==a)
			mapvera=i,aedges=&qelist[qvert[mapvera]],till=qvert[mapvera+1];
		else if(type==1 && cvsverlist[j][indexperm%size_cvs[j]]==b)
			mapverb=i;
                indexperm/=size_cvs[j];
        }
	bool flag=false;
//		anslist[ansi]=-1;
	if(aedges==NULL || mapverb==-1 || indexperm>0)
		return;
//	printf("j%d %dj",aedges[0],mapverb);
	
	for(i=0;i<till;i++){
		if(aedges[i]==mapverb){
			if(type==1){
				anslist[ansi]=-1;
				break;
			}
			flag=true;
			break;
		}
	}
	if(!flag && type==0)
		anslist[ansi]=-1;
}

__global__ void processqdnontree(int type,int a,int b,int ntans,long long int *tanslist,int *tremlist,int *qverc,int *dverc,int *qvert,int *qelist,int *dvert,int *delist,int **cvsverlist,int *size_cvs,int *qvid){
	int ansi=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ansi>=ntans)
		return ;
	long long int indexperm=tanslist[ansi];
	if(tremlist[ansi]==0)
		return;
        int mapvera,mapverb=-1;
	int *aedges=NULL,till,i;
	for(i=0;i<qverc[0];i++){
                int j=qvid[i];
		if(i==a)
			mapvera=cvsverlist[j][indexperm%size_cvs[j]],aedges=&delist[dvert[mapvera]],till=dvert[mapvera+1];
		else if(i==b)
			mapverb=cvsverlist[j][indexperm%size_cvs[j]];
                indexperm/=size_cvs[j];
        }
	
	bool flag=false;
//		anslist[ansi]=-1;
	if(aedges==NULL || mapverb==-1 || indexperm>0)
		return;
//	printf("j%d %dj",aedges[0],mapverb);
	for(i=0;i<till;i++){
		if(aedges[i]==mapverb){
			flag=true;
			break;
		}
	}
	if(!flag)
		 atomicDec((unsigned int *)&tremlist[ansi],tremlist[ansi]);
}

__device__ void process(int id,int type,int a,int b,int *qverc,int *dverc,int *qvert,int *qelist,int *dvert,int *delist,char *qverlabel,char *dverlabel,int **cvslist,int *qvid,int ** qaddnodes,int *locks,int *tempcheck,int *parent){
	int i,j;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(ver>=qverc[0])
		return ;
	
	//__syncthreads();
	//v=id;
	//for(v!=-1;v=parent[v]){
		if(atomicCAS(locks[ver],0,0xFFFFFFFF)!=0)
			return;
		for(i=qverv[v];i<qver[v+1];i++){
			if(qelist[i]!=-1 )
			while(atomicCAS(locks[qelist[i]],ver,0xFFFFFFFF)!=ver || atomicCAS(locks[qelist[i]],0,0xFFFFFFFF)!=0 );
		}
		
		
		dim3 dblocks((sqrtf(dverc[0])/16 )+ 1,(sqrtf(dverc[0])/16)+1);
		dim3 dthreads(16,16);
		findaddcvslist<<<dblocks,dthreads>>>(tempcheck,v,dvert,dverlabel,dverc,delist,qvert,qverlabel,qelist,qvid,cvslist,qaddnodes);

		atomicExch(locks[v],0);
		for(i=qver[v];i<qver[v+1];i++){
			if(qelist[i]!=-1 )
				atomicExch(locks[qelist[i]],0);
		}
	//}
}	


__global__ void findaddcvslist(bool *temp,int ver,int *d_dvert,char * d_dverlabel,int *d_dverc,int *d_delist,int *d_qvert,char *d_qverlabel,int *d_qelist,int *d_qvid,int ** d_dcvslist,int **d_addcvslist )
{
	//int i;
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	if(d_dcvslist[dver])
		return;
	if(d_dverlabel[dver]!=d_qverlabel[ver])
		return;
	int ql=d_qvert[ver+1]-d_qvert[ver];
	int dl=d_dvert[dver+1]-d_dvert[dver];
	if(ql>dl)
		return;
	//bool *checked=(bool*)malloc(sizeof(bool)*d_dverc[0]);
	//bool *checked=new bool[d_dverc[0]];
	//memset(checked,false,sizeof(bool)*d_dverc[0]);
	//chechall(bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,bool ** d_dcvslist)
	if(chechall(ver,temp,1,d_dvert[dver],d_dvert[dver+1],d_delist,d_qelist,d_qvid,d_qvert[ver],d_qvert[ver+1],d_dcvslist))
		d_addcvslist[ver][dver]=true;
	//free(checked);
}
__global__ void findaddcvs(bool *temp,int ver,int *d_dvert,char * d_dverlabel,int *d_dverc,int *d_delist,int *d_qvert,char *d_qverlabel,int *d_qelist,int *d_qvid,int ** d_dcvslist,int **d_addcvslist )
{
	//int i;
	int dver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(dver>=d_dverc[0])
		return ;
	if(d_dcvslist[dver])
		return;
	if(d_dverlabel[dver]!=d_qverlabel[ver])
		return;
	int ql=d_qvert[ver+1]-d_qvert[ver];
	int dl=d_dvert[dver+1]-d_dvert[dver];
	if(ql>dl)
		return;
	//bool *checked=(bool*)malloc(sizeof(bool)*d_dverc[0]);
	//bool *checked=new bool[d_dverc[0]];
	//memset(checked,false,sizeof(bool)*d_dverc[0]);
	//chechall(bool *check,int i,int dfrom,int dto,int *d_delist,int *d_qelist,int *d_qvid,int qfrom,int qto,bool ** d_dcvslist)
	if(chechall(ver,temp,1,d_dvert[dver],d_dvert[dver+1],d_delist,d_qelist,d_qvid,d_qvert[ver],d_qvert[ver+1],d_dcvslist))
		d_addcvslist[qvid[ver]][dver]=true;
	//free(checked);
}*//*
__global__ void doquery(int *nquery,int * type,int * vera,int * verb,int *ntans,long long int * tanslist,int *tremlist,int *nans,long long int * anslist,int ** cvsmatrix,int **cvslist,int *qverc,int *dverc,int *qvert,int *qelist,int *dvert,int *delist,int *size_cvs,int *qvid,bool *qtree,int ** cvsaddlist,int **qaddnodes,int *locks,int *tempcheck,int *parent){
	int i;	
	int qi=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	if(qi>=nquery[0])
		return ;
	//nans[0]=1;
	dim3 blocks((sqrtf(nans[0])/16 )+ 1,(sqrtf(nans[0])/16)+1);
	dim3 threads(16,16);
	if(type[qi]==0){
		int a=vera[qi];
		int b=verb[qi];
		for(i=dvert[a];i<dvert[a+1];i++){
			if(delist[i]==b)
				delist[i]=-1;
		}
		for(i=dvert[b];i<dvert[b+1];i++){
			if(delist[i]==a)
				delist[i]=-1;
		}
		processoperation<<<blocks,threads>>>(type[qi],vera[qi],verb[qi],nans[0],anslist,qverc,dverc,qvert,qelist,dvert,delist,cvslist,size_cvs,qvid);
	}
	else if(type[qi]==1){
		processoperation<<<blocks,threads>>>(type[qi],vera[qi],verb[qi],nans[0],anslist,qverc,dverc,qvert,qelist,dvert,delist,cvslist,size_cvs,qvid);
	}
	else if(type[qi]==2){
		dim3 ntblocks((sqrtf(ntans[0])/16 )+ 1,(sqrtf(ntans[0])/16)+1);
		int a=vera[qi];
		int b=verb[qi];
		for(i=qvert[a];i<qvert[a+1];i++){
			if(qelist[i]==b)
				qelist[i]=-1;
		}
		for(i=qvert[b];i<qvert[b+1];i++){
			if(qelist[i]==a)
				qelist[i]=-1;
		}
		int l=qvert[vera[qi]+1],i;
		bool flag=false,istree=false;
		for(i=qvert[vera[qi]];i<l;i++){
			if(qelist[i]==verb[qi]){
				flag=true;
				if(qtree[i])
					istree=true;
				break;
			}
		}
		if(!flag)
			return;
		if(!istree){
			processqdnontree<<<ntblocks,threads>>>(type[qi],vera[qi],verb[qi],ntans[0],tanslist,tremlist,qverc,dverc,qvert,qelist,dvert,delist,cvslist,size_cvs,qvid);
		}
		else{
			int v=a;
			while(v!=-1){
				locks[v]=v;
				v=parent[v];
			}
			//locks[ver]=ver;
			//processqdtree(qi+1,type[qi],vera[qi],verb[qi],qverc,dverc,qvert,qelist,dvert,delist,cvsmatrix,qvid,qaddnodes,locks,tempcheck,parent);
		}
	}
	else{
	}

}*/
//parms[0]=max thread size
//parms[1]=from qvertex
//parms[2]=to qvertex
//parms[3=till now size
__global__ void cperm(long long int *parms,int *d_found,int * qdmap,long long int *tillnow,int * d_qverc,int * d_qelist,int * d_qvert,int * d_dvert,int *d_delist,bool *d_qtree,int *d_size_cvs,int **d_cvsverlist,int *d_qvid){
	int i;
	int threadId=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	long long int indexperm=threadId+parms[0];
	if(parms[3]!=0)
		indexperm/=parms[3];
        for(i=parms[1]+1;i<=parms[2];i++){
                int j=d_qvid[i];
                indexperm/=d_size_cvs[j];
        }
        if(indexperm)
                return;
	if(threadId>=parms[4])
		return;
        indexperm=threadId+parms[0];
//        if(indexperm!=3409 && parms[3]!=0)
  //              return;
        d_found[threadId+1]=1;
//	int *found=&d_found[2*threadId];
	int *d_qdmap=&qdmap[d_qverc[0]*threadId];//new int[d_qverc[0]];
	if(parms[3]){
		indexperm=tillnow[indexperm%parms[3]];
		for(i=parms[1];i>=0;i--){
			int j=d_qvid[i];
			d_qdmap[i]=d_cvsverlist[j][indexperm%d_size_cvs[j]];
			//if(parms[3]!=0)
//			printf("%d ",d_qdmap[i]);
			indexperm/=d_size_cvs[j];
		}
	}
	indexperm=threadId+parms[0];
	if(parms[3]!=0)
		indexperm/=parms[3];
        for(i=parms[1]+1;i<=parms[2];i++){
                int j=d_qvid[i];
                d_qdmap[i]=d_cvsverlist[j][indexperm%d_size_cvs[j]];
//		if(parms[3]!=0)
//			printf("%d ",d_qdmap[i]);
                indexperm/=d_size_cvs[j];
        }
	for(i=0;i<=parms[2];i++){
	int j;	
		for(j=i+1;j<=parms[2];j++){
			if(d_qdmap[i]==d_qdmap[j]){
			d_found[threadId+1]=0;
			return;
			}
		}
	}
	int n,p,j,k,flag=0,ver;
	for(ver=0;ver<=parms[2];ver++){
		int l=d_qvert[ver+1];
		int dver=d_qdmap[ver];
		
		n=d_dvert[dver+1];
		for(i=d_qvert[ver];i<l;i++)
		{
			flag=0;
			j=d_qelist[i];
			if(j>parms[2])
				continue;
			k=d_qdmap[j];
			//if(!d_qtree[i])
			//	continue;
			p=d_dvert[dver];
			for(;p<n;p++){
				if(k==d_delist[p]){
					flag=1;
					break;
				}
			}
			if(!flag){

//			if(parms[3]!=0)
//			printf("iNOK ");
				d_found[threadId+1]=0;
				return;
				/*if(d_qtree[i]){				
					found[0]=found[1]=-1;		
					return;
				}
				else
					found[1]=0,found[0]++;*/
			}
		}
	}
}
__global__ void puttoanswer(long long int *parms,int *d_qvid,int *d_size_cvs,int *found,long long int *till,long long int *next){
	int i;
        int threadId=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
        long long int indexperm=threadId+parms[0];
        if(parms[3]!=0)
                indexperm/=parms[3];
        for(i=parms[1]+1;i<=parms[2];i++){
                int j=d_qvid[i];
                indexperm/=d_size_cvs[j];
        }
        if(indexperm)
                return;
	if(threadId>=parms[4])
		return;
	if(found[threadId+1]==found[threadId+2])
		return;
//	if(parms[3]!=0)
//	printf("Thread%d",threadId);
	long long int Id=threadId+parms[0];
        indexperm=0;
	if(parms[3]!=0){
        	indexperm=till[Id%parms[3]];
		Id/=parms[3];
	}
        for(i=parms[1]+1;i<=parms[2];i++){
                int j=d_qvid[i];
                indexperm*=d_size_cvs[j];
		indexperm+=Id%d_size_cvs[j];
		Id/=d_size_cvs[j];		
        }
	next[found[threadId+1]]=indexperm;
	//printf("a%llda",next[0]);	
}



#define maxthreadsize 10000000
int *h_qvert,*h_qelist,*h_dvert,*h_delist;//,*h_dvelist,*h_qvelist;
char *h_qverlabel,*d_qverlabel,*h_dverlabel,*d_dverlabel;
int **h_cvslist,**d_cvslist,**h_tem;
int **d_cvsverlist,**d_temverlist;
long long int *h_parms,*d_parms;
long long int *d_tillnow,*d_next;
int *d_found,*h_found;
void callforallperm(int i,int till,int tillnowsize,int qver,long long int mapid){
	int j,k,l;
	l=h_qvid[i-1];
	//printf("mm%lld\n",mapid);
	//printf("i%d %di",i,l);
	if(i==qver+1){
			long long int ansc=0,fix=maxthreadsize/qver/10;
			dim3 blocks((sqrt(fix)/16 )+ 1,(sqrt(fix)/16)+1);
			dim3 threads(16,16);
			for(int ii=0;(ii)*fix<mapid;ii++){
				h_parms[0]=ii*fix;
				h_parms[1]=till;
				h_parms[2]=i-2;
				h_parms[3]=tillnowsize;	
				h_parms[4]=fix;
				hipMemset(d_found,0,sizeof(int)*(fix+2));
				hipMemcpy(d_parms, h_parms, sizeof(long long int)*5, hipMemcpyHostToDevice) ;
//				printf("aaa%d %d %d %daaa",h_parms[1],h_parms[2],h_parms[3],fix);
				cperm<<<blocks,threads>>>(d_parms,d_found,d_qdmap,d_tillnow,d_qverc,d_qelist,d_qvert,d_dvert,d_delist,d_qtree,d_size_cvs,d_cvsverlist,d_qvid);
				hipMemcpy(h_found, d_found, sizeof(int)*(fix+2), hipMemcpyDeviceToHost) ;
				h_found[0]=ansc;
				thrust::exclusive_scan(h_found,h_found+fix+2,h_found);
				ansc=h_found[fix+1];
				hipMemcpy(d_found, h_found, sizeof(int)*(fix+2), hipMemcpyHostToDevice) ;
				puttoanswer<<<blocks,threads>>>(d_parms,d_qvid,d_size_cvs,d_found,d_tillnow,d_next);
//				printf("bb%lldbb\n",ansc);
			}
			mapid=tillnowsize=ansc;
			till=i-2;
			long long int * te=d_next;
			d_next=d_tillnow;
			d_tillnow=te;
			if(mapid==0)
				return;
			ansi=mapid;	
	}
	else{
		if(mapid*h_size_cvs[l]>maxthreadsize/qver && i>2){		
			long long int ansc=0,fix=maxthreadsize/qver;
			dim3 blocks((sqrt(fix)/16 )+ 1,(sqrt(fix)/16)+1);
			dim3 threads(16,16);
			for(int ii=0;(ii)*fix<mapid;ii++){
				h_parms[0]=ii*fix;
				h_parms[1]=till;
				h_parms[2]=i-2;
				h_parms[3]=tillnowsize;	
				h_parms[4]=fix;
				hipMemset(d_found,0,sizeof(int)*(fix+2));
				hipMemcpy(d_parms, h_parms, sizeof(long long int)*5, hipMemcpyHostToDevice) ;
//				printf("aaa%d %d %d %daaa",h_parms[1],h_parms[2],h_parms[3],fix);
				cperm<<<blocks,threads>>>(d_parms,d_found,d_qdmap,d_tillnow,d_qverc,d_qelist,d_qvert,d_dvert,d_delist,d_qtree,d_size_cvs,d_cvsverlist,d_qvid);
				hipMemcpy(h_found, d_found, sizeof(int)*(fix+2), hipMemcpyDeviceToHost) ;
				h_found[0]=ansc;
				thrust::exclusive_scan(h_found,h_found+fix+2,h_found);
				ansc=h_found[fix+1];
				hipMemcpy(d_found, h_found, sizeof(int)*(fix+2), hipMemcpyHostToDevice) ;
				puttoanswer<<<blocks,threads>>>(d_parms,d_qvid,d_size_cvs,d_found,d_tillnow,d_next);
//				printf("bb%lldbb\n",ansc);
	//			break;
			}
			mapid=tillnowsize=ansc;
			till=i-2;
			long long int * te=d_next;
			d_next=d_tillnow;
			d_tillnow=te;
			hipMemcpy(h_found, d_tillnow+ansc-2, sizeof(long long int)*(4), hipMemcpyDeviceToHost) ;
//			printf("zz%lld %lldzz\n",h_found[0],h_found[2]);
			if(mapid==0)
				return;
//			if(i==10)
//				return;
		}
		callforallperm(i+1,till,tillnowsize,qver,mapid*h_size_cvs[l]);

		/*for(j=0;j<dmax;j++){
				//printf("%d %d %d\n",j,check[j],cvslist[l][j]);
				if(cvslist[l][j] && !check[j]){
				//ansi++;
				check[j]=true;
				qdmap[i-1]=j;
				//mapid+=l*h_size_cvs[l];
				callforallperm(check,cvslist,i+1,max,dmax,mapid*h_size_cvs[l] +j );
				check[j]=false;
			}
		}*/
	}
}

int main(int argc, char **argv)
{
	int deviceId = 4;
	hipSetDevice(deviceId);
	int h_qverc,h_dverc;
	
	
	
	int *d_hash1,*d_hash2;
	
	int i,j;
	scanf("%d",&h_qverc);
	h_qvert=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_qvid=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_qidtov=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_tem=(int **)malloc(sizeof(int*)*(h_qverc+1));
	h_cvslist=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		scanf("%d ",&h_qvert[i]);
	}
	h_qverlabel=(char *)malloc(sizeof(char)*(h_qverc+1));
	for(i=0;i<h_qverc;i++){
		scanf("%c ",&h_qverlabel[i]);
		printf("i%ci ",h_qverlabel[i]);
	}
	printf("\n");	
	h_qelist=(int *)malloc(sizeof(int)*h_qvert[h_qverc]);
	for(i=0;i<h_qvert[h_qverc];i++)
		scanf("%d",&h_qelist[i]);
	h_qtree=(bool *)malloc(sizeof(bool)*h_qvert[h_qverc]);
	for(i=0;i<h_qvert[h_qverc];i++){
		scanf("%d",&j);
		if(j==1)
			h_qtree[i]=true;
		else
			h_qtree[i]=false;
	}
	
	scanf("%d",&h_dverc);
	h_dvert=(int *)malloc(sizeof(int)*(h_dverc+1));
	for(i=0;i<=h_dverc;i++){
		scanf("%d ",&h_dvert[i]);
	}
	h_dverlabel=(char *)malloc(sizeof(int)*(h_dverc+1));
	for(i=0;i<h_dverc;i++){
		scanf("%c ",&h_dverlabel[i]);
	}
	for(i=0;i<=h_qverc;i++)
		h_cvslist[i]=(int *)malloc(sizeof(int)*(h_dverc+1));
	
	h_delist=(int *)malloc(sizeof(int)*h_dvert[h_dverc]);
	for(i=0;i<h_dvert[h_dverc];i++)
		scanf("%d",&h_delist[i]);
	printf("Start processing\n");
	hipMalloc(&d_qverc,sizeof(int));
	hipMalloc(&d_over,sizeof(bool));
	hipMalloc(&d_qvert,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qverlabel,sizeof(char)*(h_qverc+1));
	hipMalloc(&d_qidtov,sizeof(int)*(h_qverc+1));
	//hipMalloc(&d_loc,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qelist,sizeof(int)*h_qvert[h_qverc]);
	hipMalloc(&d_qtree,sizeof(bool)*h_qvert[h_qverc]);
	hipMalloc(&d_hash1,sizeof(int)*1000038);
	hipMalloc(&d_hash2,sizeof(int)*1000038);
	
	hipMalloc(&d_qvid,sizeof(int)*(h_qverc+1));
	hipMemcpy(d_qverc,&h_qverc,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_qvert,h_qvert,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_qverlabel,h_qverlabel,sizeof(char)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_qelist,h_qelist,sizeof(int)*h_qvert[h_qverc],hipMemcpyHostToDevice);
	hipMemcpy(d_qtree,h_qtree,sizeof(bool)*h_qvert[h_qverc],hipMemcpyHostToDevice);
	hipMemset(d_hash1,0,sizeof(int)*1000038);
	//hipMemset(d_hash2,0,sizeof(int)*1000038);
	//hipMemset(d_loc,0,sizeof(int)*(h_qverc+1));
	int *qparent;
	hipMalloc(&qparent,sizeof(int)*(h_qverc+1));
	hipMemset(qparent,-1,sizeof(int)*(h_qverc+1));
	hipMemset(d_qidtov,-1,sizeof(int)*(h_qverc+1));
	hipMemset(d_qvid,0,sizeof(int)*(h_qverc+1));
	int *h_hash1=(int *)malloc(sizeof(int)*1000038);
	int *h_hash2=(int *)malloc(sizeof(int)*1000038);
	dim3 blocks((sqrt(h_qverc)/16 )+ 1,(sqrt(h_qverc)/16)+1);
	dim3 threads(16,16);
	
	//int *d_qvert,int *d_dverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_hash1,bool *d_hash2)
	h_over=true;
	//h_qvid[1]=1;
	//h_qvid[3]=1;
	//hipMemcpy(d_qvid,h_qvid,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
//printf("qt%d %dqt\n",h_qtree[0],h_qtree[1]);
	//setdeg1<<<blocks,threads>>>(d_qvert,d_qverc,d_qvid,d_qtree);

	h_over=false;
	int maxval=1;
	while(!h_over)
	{
		h_over=true;
		hipMemcpy(d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice) ;
		hipMemset(d_hash1,0,sizeof(int)*1000038);
		findhash <<<blocks,threads>>> (d_qvert,d_qverlabel,d_qverc,d_qvid,d_qelist,d_over,d_qtree,d_hash1,d_hash2);
		//(int *d_qvert,int *d_dverc,int *d_qvid,int *d_qelist,bool *d_over,bool *d_hash0,bool *d_qtree,bool *d_hash2)
		hipError_t err = hipGetLastError();
		if(err!=hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(err));
			printf("Not Ok");
		}
		hipMemcpy(h_hash1,d_hash1,sizeof(int)*1000038,hipMemcpyDeviceToHost);
		h_hash1[0]+=maxval;
		thrust::exclusive_scan(h_hash1,h_hash1+1000038,h_hash1);
		maxval=h_hash1[1000037];
		hipMemcpy(d_hash1,h_hash1,sizeof(int)*1000038,hipMemcpyHostToDevice);
		puttoid<<<blocks,threads>>>(d_qvert,d_qverlabel,d_qverc,d_qvid,d_qelist,d_qtree,d_hash1,d_qidtov,qparent);
///		hipMemcpy(h_hash2,d_hash2,sizeof(bool)*1000038,hipMemcpyDeviceToHost);
		hipMemcpy(&h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
		
		//printf("over flag:%d ",h_over);
		/*for(i=0;i<h_qverc;i++){
			//if()
				printf("%d ",h_qvid[i]);
//			if(h_hash2[i])
//				printf("h2 %d ",i);
//			if(h_hash1[i] || h_hash2[i])
//				printf("\n");
		}
		printf("\n");*/
		printf("Step %d\n",maxval);
	}	
	hipMemcpy(h_qvid,d_qvid,sizeof(int)*h_qverc,hipMemcpyDeviceToHost);
	hipMemcpy(h_qidtov,d_qidtov,sizeof(int)*(h_qverc+1),hipMemcpyDeviceToHost);
	for(i=0;i<=h_qverc;i++){
		printf("%d ",h_qidtov[i]);
	}
	printf("\n");
	for(i=0;i<=h_qverc;i++){
		printf("%d ",h_qvid[i]);
	}
	printf("\n");
	hipFree(d_qtree);
	hipFree(d_hash1);
	hipFree(d_hash2);
	free(h_hash1);
	free(h_hash2);
	free(h_qtree);
	
	hipMalloc(&d_cvslist,sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&h_tem[i],sizeof(int)*(h_dverc+1));
		hipMemset(h_tem[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemset(h_tem[1],0,sizeof(int)*(h_dverc+1));
	hipMemcpy(d_cvslist,h_tem,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMalloc(&d_dvert,sizeof(int)*(h_dverc+1));
	hipMalloc(&d_dverlabel,sizeof(char)*(h_dverc+1));
	hipMalloc(&d_dverc,sizeof(int));
	hipMalloc(&d_delist,sizeof(int)*h_dvert[h_dverc]);	
	hipMemcpy(d_dverc,&h_dverc,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_dvert,h_dvert,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_dverlabel,h_dverlabel,sizeof(char)*(h_dverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_delist,h_delist,sizeof(int)*h_dvert[h_dverc],hipMemcpyHostToDevice);
	dim3 dblocks((sqrt(h_dverc)/16 )+ 1,(sqrt(h_dverc)/16)+1);
	dim3 dthreads(16,16);
	memset(h_cvslist[1],1,sizeof(int)*(h_dverc+1));
	h_size_cvs=(int *)malloc(sizeof(int)*(h_qverc+1));
	memset(h_size_cvs,0,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_size_cvs,sizeof(int)*(h_qverc+1));
	hipMemset(d_size_cvs,0,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_cvsverlist,sizeof(int*)*(h_qverc+1));
	d_temverlist=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&d_temverlist[i],sizeof(int)*(h_dverc+1));
		hipMemset(d_temverlist[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemcpy(d_cvsverlist,d_temverlist,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);
	long long int totalthreads=1;
	int *h_temploc;
	h_temploc=(int *)malloc(sizeof(int)*(h_dverc+1));
	for(i=0;i<h_dverc;i++)
		h_temploc[i]=i;
	hipMemcpy(d_temverlist[1],h_temploc,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
	h_size_cvs[1]=h_dverc;
	bool *d_tempcheck;	
	hipMalloc(&d_tempcheck,sizeof(bool)*(h_dvert[h_dverc]+1));	
	printf("Starting cvs find\n");
	for(i=0;i<=h_qverc;i++)
	{
		if(h_qidtov[i]!=-1)
		{
		hipMemset(d_tempcheck,false,sizeof(bool)*(h_dvert[h_dverc]+1));	

		//findcvs(int ver,int *d_dvert,int *d_dverc,int *d_delist,int *d_qvert,int *d_qelist,int *d_qvid,bool ** d_dcvslist )
		findcvs<<<dblocks,dthreads>>>(d_tempcheck,h_qidtov[i],d_dvert,d_dverlabel,d_dverc,d_delist,d_qvert,d_qverlabel,d_qelist,d_qvid,d_cvslist);
		printf("id %d \n",i);
		hipMemcpy(h_cvslist[i],h_tem[i],sizeof(int)*(h_dverc+1),hipMemcpyDeviceToHost);
		//printf("%d ",h_qidtov[i]);
		thrust::exclusive_scan(h_cvslist[i],h_cvslist[i]+h_dverc+1,h_temploc);
		h_size_cvs[i]=h_temploc[h_dverc];
		hipMemcpy(h_tem[0],h_temploc,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
		puttolist<<<dblocks,dthreads>>>(d_dverc,h_tem[0],d_temverlist[i]);
		for(j=0;j<h_dverc;j++)
			if(h_cvslist[i][j])
				printf("%d ",j);
		//printf("\n");
	//	hipMemcpy(h_cvslist[i],d_temverlist[i],sizeof(int)*(h_dverc+1),hipMemcpyDeviceToHost);
		//hipMemcpy(h_temploc,h_tem[i],sizeof(int)*(h_size_cvs[i]),hipMemcpyDeviceToHost);
		//printf("On list");
		//for(j=0;j<h_size_cvs[i];j++)
		//	printf("%d ",h_temploc[j]);
		//printf("\n");	
		printf("size %d\n",h_size_cvs[i]);
		}	
		
	}
	hipMemcpy(d_size_cvs,h_size_cvs,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
	//hipMemcpy(h_delist,d_delist,sizeof(int)*(h_dvert[h_dverc]),hipMemcpyDeviceToHost);
	//	for(j=0;j<h_dvert[h_dverc];j++)
	//		printf("%d ",h_delist[j]);	
	//hipFree(d_tempcheck);
	free(h_temploc);
	bool * check=(bool *)malloc(sizeof(bool)*(h_dverc+1));
	memset(check,false,sizeof(bool)*(h_dverc+1));
	qdmap=(int *)malloc(sizeof(int)*(h_qverc+1));
	hipMalloc(&d_qdmap,sizeof(int)*(h_qverc+1));
	h_anslist=(long long int *)malloc(sizeof(long long int)*1000001);
	hipMalloc(&d_anslist,sizeof(long long int)*(1000001));
	h_treeanslist=(long long int*)malloc(sizeof(long long int)*1000001);
	hipMalloc(&d_treeanslist,sizeof(long long int)*(1000001));
	h_treeremlist=(int *)malloc(sizeof(int)*1000001);
	hipMalloc(&d_treeremlist,sizeof(int)*(1000001));
	ansi=0;
	h_parms=(long long int *)malloc(sizeof(long long int)*5);
	hipMalloc(&d_parms,sizeof(long long int)*5);
	hipMalloc(&d_tillnow,sizeof(long long int)*maxthreadsize);
	hipMalloc(&d_next,sizeof(long long int)*maxthreadsize);
	hipMalloc(&d_qdmap,sizeof(int)*maxthreadsize);
	h_found=(int *)malloc(sizeof(int)*maxthreadsize);
	hipMalloc(&d_found,sizeof(int)*maxthreadsize);
	
	callforallperm(1,-1,0,h_qverc,1);
	
	printf("Final:%d\n",ansi);
	//answers found
	/*int * d_ansi,*d_treeansi;
	hipMalloc(&d_ansi,sizeof(int));
	hipMemcpy(d_ansi,&ansi,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_anslist,h_anslist,sizeof(long long int)*(ansi),hipMemcpyHostToDevice);
	hipMalloc(&d_treeansi,sizeof(int));
	hipMemcpy(d_treeansi,&treeansi,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_treeanslist,h_treeanslist,sizeof(long long int)*(treeansi),hipMemcpyHostToDevice);
	hipMemcpy(d_treeremlist,h_treeremlist,sizeof(int)*(treeansi),hipMemcpyHostToDevice);
	

	int nqueries,*d_nqueries;
	int *h_vera,*h_verb;
	int *d_vera,*d_verb;
	int *h_type,*d_type;
	scanf("%d",&nqueries);	
	h_vera=(int*) malloc(sizeof(int)*nqueries);
	h_verb=(int*) malloc(sizeof(int)*nqueries);
	h_type=(int *) malloc(sizeof(int)*nqueries);
	
		
	hipMalloc(&d_vera,sizeof(int)*nqueries);
	hipMalloc(&d_verb,sizeof(int)*nqueries);
	hipMalloc(&d_type,sizeof(int)*nqueries);
	
	int *h_qvertadd,*h_qelistadd,*h_dvertadd,*h_delistadd;//,*h_dvelist,*h_qvelist;
	int *d_qvertadd,*d_qelistadd,*d_dvertadd,*d_delistadd;//,*h_dvelist,*h_qvelist;
	h_qvertadd=(int *)malloc(sizeof(int)*(h_qverc+1));
	h_dvertadd=(int *)malloc(sizeof(int)*(h_dverc+1));
	memset(h_qvertadd,0,sizeof(int)*(h_qverc+1));
	memset(h_dvertadd,0,sizeof(int)*(h_dverc+1));
	map<int,vector<int>> qaddlist,daddlist;
	for(i=0;i<nqueries;i++){
		scanf("%d%d%d",&h_type[i],&h_vera[i],&h_verb[i]);
		if(h_type[i]==2)
			h_qvertadd[h_vera[i]]++,qaddlist[h_vera].push_back(verb);
		else if(h_type[i]==0)
			h_dvertadd[h_vera[i]]++,daddlist[h_vera].push_back(verb);
	}
	
	thrust::exclusive_scan(h_qvertadd,h_qvertadd+h_qverc+1,h_qvertadd);
	thrust::exclusive_scan(h_dvertadd,h_dvertadd+h_dverc+1,h_dvertadd);
	
	h_qelistadd=(int *)malloc(sizeof(int)*(h_qvertadd[h_qverc]+1));
	h_delistadd=(int *)malloc(sizeof(int)*(h_dvertadd[h_dverc]+1));
	
	hipMalloc(&d_qvertadd,sizeof(int)*(h_qverc+1));
	hipMalloc(&d_dvertadd,sizeof(int)*(h_dverc+1));
	hipMalloc(&d_qelistadd,sizeof(int)*(h_qvertadd[h_qverc]+1));
	hipMalloc(&d_delistadd,sizeof(int)*(h_dvertadd[h_dverc]+1));
	
	hipMemcpy(d_qvertadd,h_qvertadd,sizeof(int)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMemcpy(d_dvertadd,h_dvertadd,sizeof(int)*(h_dverc+1),hipMemcpyHostToDevice);
	
	for(i=0;i<h_qverc;i++)
		for(j=0;j<qaddlist[i].size();j++)
			h_qelistadd[h_qvertadd[i+j]]=qaddlist[i][j];
	
	for(i=0;i<h_dverc;i++)
		for(j=0;j<daddlist[i].size();j++)
			h_delistadd[h_dvertadd[i+j]]=daddlist[i][j];
	
	hipMemcpy(d_qelistadd,h_qelistadd,sizeof(int)*(h_qvertadd[h_qverc]+1),hipMemcpyHostToDevice);
	hipMemcpy(d_delistadd,h_delistadd,sizeof(int)*(h_dvertadd[h_dverc]+1),hipMemcpyHostToDevice);
	
	hipMemcpy(d_vera,h_vera,sizeof(int)*nqueries,hipMemcpyHostToDevice);
	hipMemcpy(d_verb,h_verb,sizeof(int)*nqueries,hipMemcpyHostToDevice);
	hipMemcpy(d_type,h_type,sizeof(int)*nqueries,hipMemcpyHostToDevice);
	
	hipMalloc(&d_nqueries,sizeof(int));
	hipMemcpy(d_nqueries,&nqueries,sizeof(int),hipMemcpyHostToDevice);
	
	dim3 qblocks((sqrt(nqueries)/16 )+ 1,(sqrt(nqueries)/16)+1);
	dim3 qthreads(16,16);
	
	int **cvsaddlist,**qaddnodes,**h_cvsaddlist,**h_qaddnodes;
	int *locks;
	
	hipMalloc(&qaddnodes,sizeof(int*)*(h_qverc+1));
	h_qaddnodes=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&h_qaddnodes[i],sizeof(int)*(h_dverc+1));
		hipMemset(h_qaddnodes[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemcpy(qaddnodes,h_qaddnodes,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);

	hipMalloc(&cvsaddlist,sizeof(int*)*(h_qverc+1));
	h_cvsaddlist=(int **)malloc(sizeof(int*)*(h_qverc+1));
	for(i=0;i<=h_qverc;i++){
		hipMalloc(&h_cvsaddlist[i],sizeof(int)*(h_dverc+1));
		hipMemset(h_cvsaddlist[i],0,sizeof(int)*(h_dverc+1));
	}
	hipMemcpy(cvsaddlist,h_cvsaddlist,sizeof(int*)*(h_qverc+1),hipMemcpyHostToDevice);
	hipMalloc(&locks,sizeof(int)*(h_qverc+1));
	hipMemset(locks,0,sizeof(int)*(h_qverc+1));
	
	doquery<<<qblocks,qthreads>>>(d_nqueries,d_type,d_vera,d_verb,d_treeansi,d_treeanslist,d_treeremlist,d_ansi,d_anslist,d_cvslist,d_cvsverlist,d_qverc,d_dverc,d_qvert,d_qelist,d_dvert,d_delist,d_size_cvs,d_qvid,d_qtree,cvsaddlist,qaddnodes,locks,d_tempcheck,qparent);

	dohard<<<blocks,threads>>>(d_cvslist,d_cvsverlist,d_qverc,d_dverc,d_qvert,d_qelist,d_dvert,d_delist,d_size_cvs,d_qvid,d_qtree,cvsaddlist,qaddnodes,locks,d_qvertadd,d_qelistadd,d_dvertadd,d_delistadd);
	

	hipMemcpy(h_anslist,d_anslist,sizeof(long long int)*(ansi),hipMemcpyDeviceToHost);
	for(i=0;i<ansi;i++)
		if(h_anslist[i]==-1)
			printf(" %d ",i);
	*/hipFree(d_over);
	hipFree(d_qverc);
	hipFree(d_qvert);
	hipFree(d_qelist);
	hipFree(d_qvid);
	hipFree(d_qidtov);
	hipFree(d_dvert);
	hipFree(d_delist);
	hipFree(d_dverc);
	hipFree(d_cvslist);
	hipFree(d_cvsverlist);
	hipFree(d_size_cvs);
	hipFree(d_anslist);
	/*free(h_qvid);
	free(h_qvert);
	//free(h_qelist);
	free(h_qidtov);
	free(h_cvslist);
	free(h_dvert);
	free(h_delist);*/
}	
