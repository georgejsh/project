#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include ""
#include <thrust/scan.h>
/*
__global__ void dfs_parallel(int *d_frontier1,int *d_vertex,int *d_loc,int *d_edge,int *d_frontier2)
{
	int i = blockIdx.x * gridDim.y + blockIdx.y,j=i*blockDim.x*blockDim.y+threadIdx.x*blockDim.y+threadIdx.y;
	if(d_frontier1[j]==-1)
		return;
	int ver=d_frontier1[j],k=d_loc[j],l=d_vertex[ver+1];
    for(i=d_vertex[ver];i<l;i++)
	{
		d_frontier2[k++]=d_edge[i];
	}
}

__global__ void cull(int *d_frontier1,int *d_dist,int *d_frontier2,int *d_count)
{
	int i = blockIdx.x * gridDim.y + blockIdx.y,j=i*blockDim.x*blockDim.y+threadIdx.x*blockDim.y+threadIdx.y;
	int ver=d_frontier1[j];
	if(d_dist[ver]!=-1)
		return;
	//atomicAdd(d_count,1);
}

__global__ void findloc(int *d_front,int *d_vertex,int *d_loc)
{
	int i = blockIdx.x * gridDim.y + blockIdx.y,j=i*blockDim.x*blockDim.y+threadIdx.x*blockDim.y+threadIdx.y;
	int ver=d_front[j];
	d_loc[j]=d_vertex[ver+1]-d_vertex[ver];
}
*/
__global__ void parallel(int *d_vertex,int *d_verc,int *d_edge,int *d_dist,bool *d_over,int *d_weight)
{
	int i;
	int ver=threadIdx.x*blockDim.y+threadIdx.y+blockDim.x*blockDim.y*(blockIdx.x*gridDim.y+blockIdx.y);
	//threadIdx.x*blockIdx.y+threadIdx.y;
	//d_dist[1]=3;
	//int ver=(threadIdx.x+blockIdx.x*blockDim.x)*gridDim.y*blockDim.y+threadIdx.y+blockIdx.y*blockDim.y;
	if(ver>=d_verc[0])
		return ;
	int k=d_dist[ver];
	if(k!=-1)
	{
		int l=d_vertex[ver+1];
		for(i=d_vertex[ver];i<l;i++)
		{
			int m=d_edge[i],p=d_dist[m],q=k+d_weight[i];
			if(p==-1)
			{
				d_dist[m]=q;d_over[0]=true;
			}
			else if(p>q)
			{
				d_dist[m]=q;d_over[0]=true;
			}

		}
	}

}


	void dfs(int *d_vertex,int *d_verc,int *d_edge,int *d_dist,int *d_source)
{
	
/*
	int * d_frontier1,* d_frontier2,*d_loc,d_frontcount,done=0;
	int *temp,numBytes=sizeof(int)*d_vertex[*d_verc];
	hipMalloc(&d_frontier1,numBytes);
	hipMalloc(&d_loc,numBytes);
	hipMalloc(&d_frontier2,numBytes);

	d_frontier1[0]=*d_source;
	d_frontcount=1;


	while(d_frontcount)
	{
		dim3 blocks(d_frontcount/16,d_frontcount/16);
		dim3 threads(16, 16);
		findloc <<< blocks,threads >>> (d_frontier1,d_vertex,d_loc);
		thrust::exclusive_scan(d_loc,d_loc+d_frontcount,d_loc);

		dfs_parallel<<<blocks,threads>>>(d_frontier1,d_vertex,d_loc,d_edge,d_frontier2);
		hipMemset(d_frontier1,-1,numBytes);
		int l=d_frontier1[d_frontcount-1],k=d_loc[d_frontcount-1]+d_vertex[l+1]-d_vertex[l];
		d_frontcount=0;
		
		dim3 blocks1(k/16,k/16);
		dim3 threads1(16, 16);		
		cull<<<blocks1,threads1>>>(d_frontier2,d_dist,d_frontier1,&d_frontcount);
	}


	*/



}
int main(int argc, char **argv)
{
	
//	GpuTimer timer2;
	int i,j,k,numBytes;
	int *d_vertex,*d_verc,*d_edge, *d_dist,*d_source,*d_weight;
	int *h_vertex, h_verc,*h_edge, *h_dist, h_source,h_ec,*h_final,*h_weight;
	scanf("%d%d",&h_verc,&h_ec);
	numBytes=sizeof(int)*(h_verc+1);
	h_vertex=(int *)malloc(numBytes);
	h_dist=(int *)malloc(numBytes);
	h_final=(int *)malloc(numBytes);
	memset(h_dist,-1,numBytes);
	h_edge=(int *)malloc(sizeof(int)*h_ec);
	h_weight=(int *)malloc(sizeof(int)*h_ec);
	for(i=0;i<=h_verc;i++)
		scanf("%d",&h_vertex[i]);
	for(i=0;i<h_ec;i++)
		scanf("%d",&h_edge[i]);
	for(i=0;i<h_ec;i++)
		scanf("%d",&h_weight[i]);
	scanf("%d",&h_source);
	h_dist[h_source]=0;

//	timer2.Start();

	hipMalloc(&d_vertex,numBytes);
	hipMalloc(&d_edge,sizeof(int)*h_ec);
	hipMalloc(&d_weight,sizeof(int)*h_ec);
	hipMalloc(&d_dist,numBytes);
	hipMalloc(&d_verc,sizeof(int));
	hipMalloc(&d_source,sizeof(int));
	hipMemcpy(d_vertex,h_vertex,numBytes,hipMemcpyHostToDevice);
	hipMemcpy(d_edge,h_edge,sizeof(int)*h_ec,hipMemcpyHostToDevice);
	hipMemcpy(d_weight,h_weight,sizeof(int)*h_ec,hipMemcpyHostToDevice);
	hipMemcpy(d_verc,&h_verc,sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(d_source,&h_source,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_dist,h_dist,numBytes,hipMemcpyHostToDevice);
	
//	timer2.Stop();
//	printf("(Memory Transfer)Time Elapsed :%lfms\n",timer2.Elapsed());
//	timer2.Start();
	//dfs(d_vertex,d_verc,d_edge,d_dist,d_source);

	
	
	
	bool stop=true;
	bool *d_over;
	dim3 blocks(h_verc/1024+1,h_verc/1024+1);
	dim3 threads(32,32);
	hipMalloc(&d_over,sizeof(bool));
	while(stop)
	{
		stop=false;
		hipMemcpy(d_over, &stop, sizeof(bool), hipMemcpyHostToDevice) ;
		parallel<<<blocks,threads>>>(d_vertex,d_verc,d_edge,d_dist,d_over,d_weight);
		hipError_t err = hipGetLastError();
		if(err!=hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(err));
			printf("Not Ok");
		}
		hipMemcpy(&stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
		//printf("a");

	}
	
	
	
//	timer2.Stop();
//	printf("(Processing)Time Elapsed :%lfms\n",timer2.Elapsed());
//	timer2.Start();

	hipMemcpy(h_final,d_dist,numBytes,hipMemcpyDeviceToHost);
//	timer2.Stop();
//	printf("(Memory Transfer)Time Elapsed :%lfms\n",timer2.Elapsed());
	//hipMemcpy(&h_verc,d_verc,sizeof(int),hipMemcpyDeviceToHost);
	
	printf("Distance\n");
	for(i=0;i<h_verc;i++)
		printf("%d ",h_final[i]);
	printf("\n");
	getchar();
	hipFree(d_vertex);
	hipFree(d_verc);
	hipFree(d_source);
	hipFree(d_dist);
	hipFree(d_edge);
	free(h_dist);
	free(h_edge);
	free(h_vertex);
}
